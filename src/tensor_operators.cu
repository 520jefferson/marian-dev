#include "hip/hip_runtime.h"
// This file is part of the Marian toolkit.
// Marian is copyright (c) 2016 Marcin Junczys-Dowmunt.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include "tensor_operators.h"

namespace marian {

// @TODO: handle this better, maybe per thread?
static hipblasHandle_t create_handle() {
  hipblasHandle_t cublasHandle;
  hipblasCreate(&cublasHandle);
  return cublasHandle;
}

static hipdnnHandle_t create_handle_dnn() {
  hipdnnHandle_t cudnnHandle;
  hipdnnCreate(&cudnnHandle);
  return cudnnHandle;
}

hipblasHandle_t cublasHandle = create_handle();
hipdnnHandle_t cudnnHandle = create_handle_dnn();

void CudnnSoftmax(Tensor out, Tensor in) {
    float alpha = 1, beta = 0;
    auto inGpu = static_cast<TensorGPU*>(in.get());
    auto outGpu = static_cast<TensorGPU*>(out.get());
    hipdnnSoftmaxForward(cudnnHandle,
                        HIPDNN_SOFTMAX_ACCURATE,
                        HIPDNN_SOFTMAX_MODE_CHANNEL,
                        &alpha,
                        inGpu->cudnn(),
                        inGpu->data(),
                        &beta,
                        outGpu->cudnn(),
                        outGpu->data());
    hipDeviceSynchronize();
}

void CudnnLogSoftmax(Tensor out, Tensor in) {
    float alpha = 1, beta = 0;
    auto inGpu = static_cast<TensorGPU*>(in.get());
    auto outGpu = static_cast<TensorGPU*>(out.get());
    hipdnnSoftmaxForward(cudnnHandle,
                        HIPDNN_SOFTMAX_LOG,
                        HIPDNN_SOFTMAX_MODE_CHANNEL,
                        &alpha,
                        inGpu->cudnn(),
                        inGpu->data(),
                        &beta,
                        outGpu->cudnn(),
                        outGpu->data());
    hipDeviceSynchronize();
}

void CudnnSoftmaxGrad(Tensor grad, Tensor adj, Tensor val) {
    float alpha = 1, beta = 0;
    auto valGpu = static_cast<TensorGPU*>(val.get());
    auto adjGpu = static_cast<TensorGPU*>(adj.get());
    auto gradGpu = static_cast<TensorGPU*>(grad.get());
    hipdnnSoftmaxBackward(cudnnHandle,
                        HIPDNN_SOFTMAX_ACCURATE,
                        HIPDNN_SOFTMAX_MODE_CHANNEL,
                        &alpha,
                        valGpu->cudnn(),
                        valGpu->data(),
                        adjGpu->cudnn(),
                        adjGpu->data(),
                        &beta,
                        gradGpu->cudnn(),
                        gradGpu->data());
    hipDeviceSynchronize();
}

void CudnnLogSoftmaxGrad(Tensor grad, Tensor adj, Tensor val) {
    float alpha = 1, beta = 0;
    auto valGpu = static_cast<TensorGPU*>(val.get());
    auto adjGpu = static_cast<TensorGPU*>(adj.get());
    auto gradGpu = static_cast<TensorGPU*>(grad.get());
    hipdnnSoftmaxBackward(cudnnHandle,
                        HIPDNN_SOFTMAX_LOG,
                        HIPDNN_SOFTMAX_MODE_CHANNEL,
                        &alpha,
                        valGpu->cudnn(),
                        valGpu->data(),
                        adjGpu->cudnn(),
                        adjGpu->data(),
                        &beta,
                        gradGpu->cudnn(),
                        gradGpu->data());
    hipDeviceSynchronize();
}

__global__ void gSoftmax(float* out,
                         const Shape outShape,
                         const float* in,
                         const float* mask) {
  int rows = outShape[0];
  int cols = outShape[1];
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* so = out + j * cols;
      const float* sp = in + j * cols;
      const float* mp = mask ? (mask + j * cols) : 0;

      extern __shared__ float _share[];

      float* _max = _share + blockDim.x;
      _max[threadIdx.x] = sp[threadIdx.x]; // mask
      for(int tid = 1; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if (id < cols) {
          if (sp[id] > _max[threadIdx.x])
            _max[threadIdx.x] = sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if (threadIdx.x < (len >> 1)) {
          if (_max[threadIdx.x + skip] > _max[threadIdx.x]) {
             _max[threadIdx.x] = _max[threadIdx.x + skip];
          }
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float max = _max[0];
      __syncthreads();

      float* _sum = _share + blockDim.x;

      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float ex = 0;
          if(!mask || mp[id])
            ex = __expf(sp[id] - max);
          so[id] = ex;
          _sum[threadIdx.x] += ex;
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      for(int tid = 0; tid < cols; tid += blockDim.x){
        int id = tid + threadIdx.x;
        if(id < cols)
          so[id] /= _sum[0];
      }
    }
  }
}

void Softmax(Tensor out, Tensor in, Tensor mask) {
  size_t m = out->shape()[0] * out->shape()[2] * out->shape()[3];
  size_t k = out->shape()[1];

  int blocks = std::min(MAX_BLOCKS, (int) m);
  int threads = std::min(MAX_THREADS, (int) k);
  int shared = sizeof(float) * threads * 2;

  if(mask)
    gSoftmax<<<blocks, threads, shared>>>(out->data(),
                                          out->shape(),
                                          in->data(),
                                          mask->data());
  else
    gSoftmax<<<blocks, threads, shared>>>(out->data(),
                                          out->shape(),
                                          in->data(),
                                          0);
  hipStreamSynchronize(0);
}

__global__ void gLogSoftmax(float* out,
                            const Shape outShape,
                            const float* in) {
  int rows = outShape[0];
  int cols = outShape[1];
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* so = out + j * cols;
      const float* sp = in + j * cols;

      extern __shared__ float _share[];

      float* _max = _share + blockDim.x;
      _max[threadIdx.x] = sp[threadIdx.x];
      for(int tid = 1; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if (id < cols) {
          if (sp[id] > _max[threadIdx.x]) _max[threadIdx.x] = sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if (threadIdx.x < (len >> 1)) {
          if (_max[threadIdx.x + skip] > _max[threadIdx.x]) {
             _max[threadIdx.x] = _max[threadIdx.x + skip];
          }
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float max = _max[0];
      __syncthreads();

      float* _sum = _share + blockDim.x;

      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float sm = sp[id] - max;
          float ex = __expf(sm);
          so[id] = sm;
          _sum[threadIdx.x] += ex;
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      for(int tid = 0; tid < cols; tid += blockDim.x){
        int id = tid + threadIdx.x;
        if(id < cols)
          so[id] -= __logf(_sum[0]);
      }
    }
  }
}

void LogSoftmax(Tensor out, Tensor in) {
  size_t m = out->shape()[0];
  size_t k = out->shape()[1];

  int blocks = std::min(MAX_BLOCKS, (int) m);
  int threads = std::min(MAX_THREADS, (int) k);
  int shared = sizeof(float) * threads * 2;

  gLogSoftmax<<<blocks, threads, shared>>>(out->data(),
                                           out->shape(),
                                           in->data());

}


///////////////////////////////////////////////////////

__global__ void gSoftmaxGrad(float* grad, const float* adj, const float* val,
                             const int rows, const int cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      extern __shared__ float _share[];
      float* _sum = _share + blockDim.x;

      float* gradRow = grad + j * cols;
      const float* adjRow = adj + j * cols;
      const float* valRow = val + j * cols;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          _sum[threadIdx.x] += valRow[id] * adjRow[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      for(int tid = 0; tid < cols; tid += blockDim.x){
        int id = tid + threadIdx.x;
        if(id < cols)
          gradRow[id] += valRow[id] * (adjRow[id] - _sum[0]);
      }
    }
  }
}

void SoftmaxGrad(Tensor grad, Tensor adj, Tensor val) {
  // grad and val are both m-by-k matrices, passed as input.
  // A weighted average of each row of grad (according to the weights
  // specified in val) is computed and subtracted from Out.
  // adj is multiplied for each element to get backward step in autodiff
  int m = grad->shape()[0] * grad->shape()[2] * grad->shape()[3];
  int k = grad->shape()[1];

  int blocks = std::min(MAX_BLOCKS, m);
  int threads = std::min(MAX_THREADS, k);
  int shared = sizeof(float) * threads * 2;
  gSoftmaxGrad<<<blocks, threads, shared>>>(grad->data(),
                                            adj->data(),
                                            val->data(),
                                            m, k);

}

__global__ void gLogSoftmaxGrad(float* grad, const float* adj, const float* val,
                                const int rows, const int cols) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      extern __shared__ float _share[];
      float* _sum = _share + blockDim.x;

      float* gradRow = grad + j * cols;
      const float* adjRow = adj + j * cols;
      const float* valRow = val + j * cols;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          _sum[threadIdx.x] += adjRow[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();
      for(int tid = 0; tid < cols; tid += blockDim.x){
        int id = tid + threadIdx.x;
        if(id < cols)
          gradRow[id] += adjRow[id] - (expf(valRow[id]) * _sum[0]);
      }
    }
  }
}

void LogSoftmaxGrad(Tensor grad, Tensor adj, Tensor val) {
  // grad and val are both m-by-k matrices, passed as input.
  // A weighted average of each row of grad (according to the weights
  // specified in val) is computed and subtracted from Out.
  // adj is multiplied for each element to get backward step in autodiff
  int m = grad->shape()[0];
  int k = grad->shape()[1];

  int blocks = std::min(MAX_BLOCKS, m);
  int threads = std::min(MAX_THREADS, k);
  int shared = sizeof(float) * threads * 2;
  gLogSoftmaxGrad<<<blocks, threads, shared>>>(grad->data(),
                                               adj->data(), val->data(),
                                               m, k);

}

///////////////////////////////////////////////////////
__global__ void gArgmax(float *out, const float *data, size_t rows, size_t cols) {
  size_t row = blockIdx.x;
    size_t startInd = row * cols;
    float maxScore = -99999;
    size_t maxInd;
    for (size_t col = 0; col < cols; ++col) {
      size_t ind = startInd + col;
      float score = data[ind];
      if (score > maxScore) {
        maxScore = score;
        maxInd = col;
      }
    }
    out[row] = maxInd;
}

//void Argmax(Tensor* Out, const Tensor* In) {
//  size_t m = In->shape()[0];
//  size_t k = In->shape()[1];
//
//  int blocks = m; //std::min(MAX_BLOCKS, (int) m);
//  int threads = k; //std::min(MAX_THREADS, (int) k);
//  //int shared = sizeof(float) * threads * 2;
//  gArgmax<<<blocks, threads>>>(Out->data(), In->data(), m, k);
//
//}

///////////////////////////////////////////////////////

void Prod(hipblasHandle_t handle, Tensor C, const Tensor A, const Tensor B,
             bool transA, bool transB, Float beta) {
  Float alpha = 1.0;

  size_t m = A->shape()[0] * A->shape()[2] * A->shape()[3];
  size_t k = A->shape()[1];
  if(transA)
    std::swap(m, k);

  size_t l = B->shape()[0];
  size_t n = B->shape()[1];
  if(transB)
    std::swap(l, n);

  size_t lda = A->shape()[1];
  size_t ldb = B->shape()[1];
  size_t ldc = B->shape()[1];

  if(transB)
    ldc = B->shape()[0];

  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  hipblasSgemm(handle, opB, opA,
              n, m, k, &alpha, B->data(), ldb, A->data(), lda, &beta, C->data(), ldc);
}

void Prod(Tensor C, const Tensor A, const Tensor B,
            bool transA, bool transB, Float beta) {

  Prod(cublasHandle, C, A, B, transA, transB, beta);
}

void CudnnDropoutPrepare(Tensor in, float p,
                         hipdnnDropoutDescriptor_t* dropDesc,
                         void** space, size_t* spaceSize,
                         void** states, size_t seed) {
  size_t statesSize;
  hipdnnDropoutGetStatesSize(cudnnHandle, &statesSize);
  auto inGpu = static_cast<TensorGPU*>(in.get());
  cudnnDropoutGetReserveSpaceSize(inGpu->cudnn(), spaceSize);

  hipMalloc((void**)states, statesSize);
  hipMalloc((void**)space, *spaceSize);

  hipdnnCreateDropoutDescriptor(dropDesc);
  hipdnnSetDropoutDescriptor(*dropDesc,
                            cudnnHandle,
                            p,
                            (void*)*states,
                            statesSize,
                            seed);
}

void CudnnDropoutDestroy(hipdnnDropoutDescriptor_t dropDesc,
                         void* space, void* states) {
  hipdnnDestroyDropoutDescriptor(dropDesc);
  hipFree(space);
  hipFree(states);
}

void CudnnDropoutForward(hipdnnDropoutDescriptor_t dropoutDesc,
                  void* space, size_t spaceSize,
                  Tensor out, Tensor in) {
  auto inGpu = static_cast<TensorGPU*>(in.get());
  auto outGpu = static_cast<TensorGPU*>(out.get());
  cudnnDropoutForward(cudnnHandle,
                      dropoutDesc,
                      inGpu->cudnn(),
                      inGpu->data(),
                      outGpu->cudnn(),
                      outGpu->data(),
                      space,
                      spaceSize);
}

void CudnnDropoutBackward(hipdnnDropoutDescriptor_t dropoutDesc,
                          void* space, size_t spaceSize,
                          Tensor out, Tensor in) {
  auto inGpu = static_cast<TensorGPU*>(in.get());
  auto outGpu = static_cast<TensorGPU*>(out.get());
  cudnnDropoutBackward(cudnnHandle,
                      dropoutDesc,
                      inGpu->cudnn(),
                      inGpu->data(),
                      outGpu->cudnn(),
                      outGpu->data(),
                      space,
                      spaceSize);
}

__global__ void gCopyRows(float* out, const float* in, size_t cols,
                          const size_t* sourceRowIdx, size_t rows) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      size_t dstId = j;
      size_t srcId = sourceRowIdx[j];

      float* rowOut = out + dstId * cols;
      const float* rowIn = in + srcId * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols)
          rowOut[i] = rowIn[i];
      }
    }
  }
}

void CopyRows(Tensor out, const Tensor in, const DeviceVector<size_t>& indeces) {
  size_t cols = in->shape()[1];
  size_t rowsToCopy = indeces.size();

  int threads = std::min(MAX_THREADS, (int)cols);
  int blocks = std::min(MAX_BLOCKS, (int)rowsToCopy);

  gCopyRows<<<blocks, threads>>>(out->data(), in->data(), cols,
                                 thrust::raw_pointer_cast(indeces.data()),
                                 rowsToCopy);

}

__global__ void gPasteRows(float* out, const float* in, size_t cols,
                          const size_t* targetRowIdx, size_t rows) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      size_t dstId = targetRowIdx[j];
      size_t srcId = j;

      float* rowOut = out + dstId * cols;
      const float* rowIn = in + srcId * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols)
          atomicAdd(rowOut + i, rowIn[i]);
      }
    }
  }
}

void PasteRows(Tensor out, const Tensor in, const DeviceVector<size_t>& indeces) {
  size_t cols = in->shape()[1];
  size_t rowsToCopy = indeces.size();

  int threads = std::min(MAX_THREADS, (int)cols);
  int blocks = std::min(MAX_BLOCKS, (int)rowsToCopy);

  gPasteRows<<<blocks, threads>>>(out->data(), in->data(), cols,
                                  thrust::raw_pointer_cast(indeces.data()),
                                  rowsToCopy);


}

void Transpose(Tensor out, const Tensor in) {
  size_t m = in->shape()[0];
  size_t n = in->shape()[1];
  float alpha = 1.0;
  float beta  = 0.0;

  hipblasSgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, &alpha, in->data(), n,
              &beta, in->data(), n, out->data(), m);
}

void Concatenate0(Tensor out, const std::vector<Tensor>& inputs) {
  size_t offset = 0;
  for(auto in : inputs) {
    UTIL_THROW_IF2(out->shape()[1] != in->shape()[1],
                   "Second dimension must be equal");
    hipMemcpy(out->data() + offset,
               in->data(),
               in->size() * sizeof(float),
               hipMemcpyDeviceToDevice);
    offset += in->size();
  }
}

__global__ void gInsertCols(float* out, const float* in,
                            size_t rows, size_t cols,
                            size_t cols_out, size_t cols_in,
                            size_t offset_out, size_t offset_in) {
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float* rowOut = out + j * cols_out + offset_out;
      const float* rowIn = in + j * cols_in + offset_in;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols)
          rowOut[i] = rowIn[i];
      }
    }
  }
}

// this probably does not work for tensors with more than 2
// dimensions, verify this!
void Concatenate1(Tensor out, const std::vector<Tensor>& inputs) {
  size_t offset = 0;
  int rows = out->shape()[0];
  int cols_out = out->shape()[1];

  for(auto in : inputs) {
    UTIL_THROW_IF2(out->shape()[0] != in->shape()[0],
                   "First dimension must be equal");
    int cols_in = in->shape()[1];

    int blocks  = std::min(MAX_BLOCKS, rows);
    int threads = std::min(MAX_THREADS, cols_in);

    gInsertCols<<<blocks, threads>>>(
      out->data(),
      in->data(),
      rows, cols_in,
      cols_out, cols_in,
      offset, 0);
    offset += cols_in;
  }
}

void Concatenate(Tensor out, const std::vector<Tensor>& inputs, int ax) {
  if(ax == 1)
    Concatenate1(out, inputs);
  else
    Concatenate0(out, inputs);
}

void Deconcatenate0(std::vector<Tensor>& outputs, const Tensor in) {
  size_t offset = 0;
  for(auto out : outputs) {
    hipMemcpy(out->data(),
               in->data() + offset,
               out->size() * sizeof(float),
               hipMemcpyDeviceToDevice);
    offset += out->size();
  }
}

void Deconcatenate1(std::vector<Tensor>& outputs, const Tensor in) {
  size_t offset = 0;
  int rows = in->shape()[0];
  int cols_in = in->shape()[1];
  for(auto out : outputs) {
    UTIL_THROW_IF2(out->shape()[0] != in->shape()[0],
                   "First dimension must be equal");
    int cols_out = out->shape()[1];

    int blocks  = std::min(MAX_BLOCKS, rows);
    int threads = std::min(MAX_THREADS, cols_out);

    gInsertCols<<<blocks, threads>>>(
      out->data(),
      in->data(),
      rows, cols_out,
      cols_out, cols_in,
      0, offset);
    offset += cols_out;
  }
}

void Deconcatenate(std::vector<Tensor>& outputs, const Tensor in, int ax) {
  if(ax == 1)
    Deconcatenate1(outputs, in);
  else
    Deconcatenate0(outputs, in);
}

__global__ void gGRUFastForward(float* out,
                                const float* state,
                                const float* xW,
                                const float* sU,
                                const float* b,
                                const float* mask,
                                size_t rows, size_t cols,
                                bool final) {

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float m = !mask || mask[j];
      float* rowOut = out + j * cols;
      const float* rowState = state + j * cols;

      const float* xWrow = xW + j * cols * 3;
      const float* sUrow = sU + j * cols * 3;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          float ev1 = expf(-(xWrow[i] + sUrow[i] + b[i]));
          float r = 1.0f / (1.0f + ev1);

          int k = i + cols;
          float ev2 = expf(-(xWrow[k] + sUrow[k] + b[k]));
          float z = 1.0f / (1.0f + ev2);

          int l = i + 2 * cols;
          float h;
          if(final)
            h = tanhf(xWrow[l] + (sUrow[l] + b[l]) * r);
          else
            h = tanhf(xWrow[l] + sUrow[l] * r + b[l]);

          float out = (1.0f - z) * h + z * rowState[i];
          rowOut[i] = m * out + (1 - m) * rowState[i];
        }
      }
    }
  }
}

void GRUFastForward(Tensor out, const std::vector<Tensor>& inputs, bool final){
  int rows = out->shape()[0];
  int cols = out->shape()[1];

  int blocks  = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gGRUFastForward<<<blocks, threads>>>(
    out->data(), // output
    inputs[0]->data(), // state
    inputs[1]->data(), // xW
    inputs[2]->data(), // sU
    inputs[3]->data(), // b
    inputs.size() > 4 ? inputs[4]->data() : 0, // mask
    rows, cols, final);
}

__global__ void gGRUFastBackward(float* outState,
                                 float* outXW,
                                 float* outSU,
                                 float* outB,
                                 const float* state,
                                 const float* xW,
                                 const float* sU,
                                 const float* b,
                                 const float* mask,
                                 const float* adj,
                                 size_t rows, size_t cols,
                                 bool final) {

  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      float m = !mask || mask[j];

      float* rowOutState = outState + j * cols;
      float* rowOutXW = outXW + j * cols * 3;
      float* rowOutSU = outSU + j * cols * 3;

      const float* rowState = state + j * cols;
      const float* rowXW = xW + j * cols * 3;
      const float* rowSU = sU + j * cols * 3;
      const float* rowAdj = adj + j * cols;

      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int i = tid + threadIdx.x;
        if(i < cols) {
          int k = i + cols;
          int l = i + 2 * cols;

          float ev1 = expf(-(rowXW[i] + rowSU[i] + b[i]));
          float r = 1.0f / (1.0f + ev1);

          float ev2 = expf(-(rowXW[k] + rowSU[k] + b[k]));
          float z = 1.0f / (1.0f + ev2);

          float h;
          if(final)
            h = tanhf(rowXW[l] + (rowSU[l] + b[l]) * r);
          else
            h = tanhf(rowXW[l] + rowSU[l] * r + b[l]);

          float adj = rowAdj[i];

          float t = (1-z)*(1-h*h);

          // df/ds
          if(outState) rowOutState[i] += m * z * adj - m + 1;

          // df/d(xW_r) ...
          float dfdxW_r = r * (1-r) * t * adj;
          if(final)
            dfdxW_r *= rowSU[l] + b[l];
          else
            dfdxW_r *= rowSU[l];
          if(outXW) rowOutXW[i] += m * dfdxW_r;
          if(outSU) rowOutSU[i] += m * dfdxW_r;
          if(outB)  atomicAdd(outB + i, m * dfdxW_r);

          // df/d(xW_z) ...
          float dfdxW_z = (1-z)*z*(rowState[i]-h) * adj;
          if(outXW) rowOutXW[k] += m * dfdxW_z;
          if(outSU) rowOutSU[k] += m * dfdxW_z;
          if(outB)  atomicAdd(outB + k, m * dfdxW_z);

          // df/d(xW_x) ...
          float dfdxW_x = t * adj;
          if(outXW) rowOutXW[l] += m * dfdxW_x;
          if(outSU) rowOutSU[l] += m * dfdxW_x * r;
          if(final)
            if(outB) atomicAdd(outB + l, m * dfdxW_x * r);
          else
            if(outB) atomicAdd(outB + l, m * dfdxW_x);
        }
      }
    }
  }
}

void GRUFastBackward(std::vector<Tensor>& outputs,
                     const std::vector<Tensor>& inputs,
                     const Tensor adj, bool final) {
  int rows = adj->shape()[0];
  int cols = adj->shape()[1];

  int blocks  = std::min(MAX_BLOCKS, rows);
  int threads = std::min(MAX_THREADS, cols);

  gGRUFastBackward<<<blocks, threads>>>(
    outputs[0] ? outputs[0]->data() : nullptr, // state - adj
    outputs[1] ? outputs[1]->data() : nullptr, // xW - adj
    outputs[2] ? outputs[2]->data() : nullptr, // sU - adj
    outputs[3] ? outputs[3]->data() : nullptr, // b - adj
    inputs[0]->data(), // state
    inputs[1]->data(), // xW
    inputs[2]->data(), // sU
    inputs[3]->data(), // b
    inputs.size() > 4 ? inputs[4]->data() : 0, // mask
    adj->data(),
    rows, cols, final);
}

__global__ void gCrossEntropyPick(float* out,
                                  const Shape outShape,
                                  const float* in,
                                  const Shape inShape,
                                  const float* pick) {

  int rows = inShape[0];
  int cols = inShape[1];
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      const float* sp = in + j * cols;

      extern __shared__ float _share[];
      float* _max = _share + blockDim.x;

      _max[threadIdx.x] = sp[threadIdx.x];
      for(int tid = 1; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if (id < cols) {
          if (sp[id] > _max[threadIdx.x]) _max[threadIdx.x] = sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if (threadIdx.x < (len >> 1)) {
          if (_max[threadIdx.x + skip] > _max[threadIdx.x]) {
             _max[threadIdx.x] = _max[threadIdx.x + skip];
          }
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float max = _max[0];
      __syncthreads();

      float* _sum = _share + blockDim.x;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          _sum[threadIdx.x] += __expf(sp[id] - max);
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();

      // cross-entropy
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id == (int)pick[j]) {
          out[j] = __logf(_sum[0]) - sp[id] + max;
        }
      }
    }
  }
}

void CrossEntropyPick(Tensor out, Tensor in, Tensor pick) {
  size_t m = in->shape()[0];
  size_t k = in->shape()[1];

  int blocks = std::min(MAX_BLOCKS, (int) m);
  int threads = std::min(MAX_THREADS, (int) k);
  int shared = sizeof(float) * threads * 2;

  gCrossEntropyPick<<<blocks, threads, shared>>>(out->data(),
                                                 out->shape(),
                                                 in->data(),
                                                 in->shape(),
                                                 pick->data());

}

__global__ void gCrossEntropyPickBackward(float* out,
                                          const Shape outShape,
                                          const float* adj,
                                          const float* in,
                                          const float* pick) {

  int rows = outShape[0];
  int cols = outShape[1];
  for(int bid = 0; bid < rows; bid += gridDim.x) {
    int j = bid + blockIdx.x;
    if(j < rows) {
      const float* sp = in + j * cols;
      float* so = out + j * cols;

      extern __shared__ float _share[];
      float* _max = _share + blockDim.x;

      _max[threadIdx.x] = sp[threadIdx.x];
      for(int tid = 1; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if (id < cols) {
          if (sp[id] > _max[threadIdx.x]) _max[threadIdx.x] = sp[id];
        }
      }
      __syncthreads();
      int len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if (threadIdx.x < (len >> 1)) {
          if (_max[threadIdx.x + skip] > _max[threadIdx.x]) {
             _max[threadIdx.x] = _max[threadIdx.x + skip];
          }
        }
        len = (len + 1) >> 1;
      }
      __syncthreads();
      float max = _max[0];
      __syncthreads();

      float* _sum = _share + blockDim.x;
      _sum[threadIdx.x] = 0.0;
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float ex = __expf(sp[id] - max);
          _sum[threadIdx.x] += ex;
        }
      }
      __syncthreads();
      len = blockDim.x;
      while(len != 1) {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
          _sum[threadIdx.x] += _sum[threadIdx.x + skip];
        len = (len + 1) >> 1;
      }
      __syncthreads();

      // cross-entropy
      for(int tid = 0; tid < cols; tid += blockDim.x) {
        int id = tid + threadIdx.x;
        if(id < cols) {
          float sub = (float)(id == (int)pick[j]);
          so[id] += adj[j] * (__expf(sp[id] - max) / _sum[0] - sub);
        }
      }
    }
  }
}

void CrossEntropyPickBackward(Tensor out, Tensor adj, Tensor a, Tensor pick) {
  size_t m = out->shape()[0];
  size_t k = out->shape()[1];

  int blocks = std::min(MAX_BLOCKS, (int) m);
  int threads = std::min(MAX_THREADS, (int) k);
  int shared = sizeof(float) * threads * 2;

  gCrossEntropyPickBackward<<<blocks, threads, shared>>>(out->data(),
                                                         out->shape(),
                                                         adj->data(),
                                                         a->data(),
                                                         pick->data());
}


}
