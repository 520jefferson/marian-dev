
#include "marian.h"
#include "mnist.h"
#include "npz_converter.h"

using namespace marian;
using namespace keywords;

int main(int argc, char** argv) {
  
  hipSetDevice(0);
  
  const size_t IMAGE_SIZE = 784;
  const size_t LABEL_SIZE = 10;
  const size_t BATCH_SIZE = 24;
  int numofdata;

  std::cerr << "Loading test set...";
  std::vector<float> testImages = datasets::mnist::ReadImages("../examples/mnist/t10k-images-idx3-ubyte", numofdata, IMAGE_SIZE);
  std::vector<float> testLabels = datasets::mnist::ReadLabels("../examples/mnist/t10k-labels-idx1-ubyte", numofdata, LABEL_SIZE);
  std::cerr << "\tDone." << std::endl;

  std::cerr << "Loading model params...";
  NpzConverter converter("../scripts/test_model/model.npz");

  std::vector<float> wData;
  Shape wShape;
  converter.Load("weights", wData, wShape);

  std::vector<float> bData;
  Shape bShape;
  converter.Load("bias", bData, bShape);

  auto initW = [wData](Tensor t) {
    t.set(wData);
  };

  auto initB = [bData](Tensor t) {
    t.set(bData);
  };

  std::cerr << "\tDone." << std::endl;


  auto x = input(shape={whatevs, IMAGE_SIZE}, name="X");
  auto y = input(shape={whatevs, LABEL_SIZE}, name="Y");

  auto w = param(shape={IMAGE_SIZE, LABEL_SIZE}, name="W0", init=initW);
  auto b = param(shape={1, LABEL_SIZE}, name="b0", init=initB);

  std::cerr << "Building model...";
  auto predict = softmax(dot(x, w) + b, axis=1, name="pred");

  std::cerr << "Done." << std::endl;

  Tensor xt({BATCH_SIZE, IMAGE_SIZE});

  size_t acc = 0;
  size_t startId = 0;
  size_t endId = startId + BATCH_SIZE;

  while (endId < numofdata) {
    std::vector<float> tmp(testImages.begin() + (startId * IMAGE_SIZE),
                           testImages.begin() + (endId * IMAGE_SIZE));
    xt << tmp;
    x = xt;

    predict.forward(BATCH_SIZE);

    std::vector<float> results(LABEL_SIZE * BATCH_SIZE);
    results << predict.val();

    for (size_t i = 0; i < BATCH_SIZE * LABEL_SIZE; i += LABEL_SIZE) {
      size_t correct = 0;
      size_t predicted = 0;
      for (size_t j = 0; j < LABEL_SIZE; ++j) {
        if (testLabels[startId * LABEL_SIZE + i + j]) correct = j;
        if (results[i + j] > results[i + predicted]) predicted = j;
      }
      acc += (correct == predicted);
    }

    startId += BATCH_SIZE;
    endId += BATCH_SIZE;
  }
  if (endId != numofdata) {
    endId = numofdata;
    if (endId - startId > 0) {
      std::vector<float> tmp(testImages.begin() + (startId * IMAGE_SIZE),
                             testImages.begin() + (endId * IMAGE_SIZE));
      xt << tmp;
      x = xt;

      predict.forward(endId - startId);

      std::vector<float> results(LABEL_SIZE * BATCH_SIZE);
      results << predict.val();

      for (size_t i = 0; i < (endId - startId) * LABEL_SIZE; i += LABEL_SIZE) {
        size_t correct = 0;
        size_t predicted = 0;
        for (size_t j = 0; j < LABEL_SIZE; ++j) {
          if (testLabels[startId * LABEL_SIZE + i + j]) correct = j;
          if (results[i + j] > results[i + predicted]) predicted = j;
        }
        acc += (correct == predicted);
      }
    }
  }
  std::cerr << "ACC: " << float(acc)/numofdata << std::endl;

  return 0;
}
