#include "hip/hip_runtime.h"
#include "marian.h"
#include "training/rescorer.h"

int main(int argc, char** argv) {
  using namespace marian;

  auto options = New<Config>(argc, argv, true, false);

  // @TODO: these options should be set in relevant classes
  options->set<bool>("dynamic-batching", false);
  options->set<size_t>("maxi-batch", 1);
  options->set<size_t>("max-length", 1000);

  auto task = rescorerByType(options);

  boost::timer::cpu_timer timer;
  task->run();
  LOG(info, "Total time: {}", timer.format());

  return 0;
}
