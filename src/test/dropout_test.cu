#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <boost/timer/timer.hpp>
#include <boost/chrono.hpp>

#include "tensors/tensor_allocator.h"
#include "tensors/tensor.h"
#include "kernels/tensor_operators.h"

#include "layers/dropout.h"

#include "kernels/dropout_cudnn.h"

using namespace marian;

int main() {
  int cudaDevice = 2;
  TensorAllocator* params = new TensorAllocator(cudaDevice);

  hipblasHandle_t handle = create_handle(cudaDevice);

  int rows = 1000;
  int cols = 50000;
  int rep = 1000;
  const float prob = 0.05f;

  Tensor dropoutMatrix;
  params->allocate(dropoutMatrix, {rows, cols, 1});

  DropoutGenerator dropout(0);

  hipStreamSynchronize(0);
  boost::timer::cpu_timer timer;

  for (int i = 0; i < rep;++i) {
    dropout.Generate(dropoutMatrix, prob);
  }

  hipDeviceSynchronize();

  std::cerr << "DropoutGenerator: " << rep << " repetitions: " << timer.format(5, "%ws") << std::endl;

  Tensor cudnnInTensor, cudnnOutTensor;
  params->allocate(cudnnInTensor, {rows, cols, 1});
  params->allocate(cudnnOutTensor, {rows, cols, 1});

  void* states_;
  void* space_;
  size_t spaceSize_;
  hipdnnDropoutDescriptor_t dropDesc_;

  CudnnDropoutPrepare(cudnnInTensor, prob, &dropDesc_, &space_, &spaceSize_, &states_, (size_t)1234);
  hipStreamSynchronize(0);

  hipDeviceSynchronize();
  timer.start();
  for (int i = 0; i < rep; ++i) {
    CudnnDropoutForward(dropDesc_, space_, spaceSize_, cudnnInTensor, cudnnOutTensor);
  }

  hipDeviceSynchronize();
  std::cerr << "CUDNN Dropout: " << rep << " repetitions: " << timer.format(5, "%ws") << std::endl;


  return 0;
}
