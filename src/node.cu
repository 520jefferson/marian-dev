#include "hip/hip_runtime.h"
#include "node.h"
#include "tensor_operators.h"

namespace marian {

// for backward_numeric
void Node::calc_numeric_grad(
		  Float delta,
		  Tensor input,
		  Tensor grad
		  )
{
  using namespace std;

  size_t inputSize = GetTotalSize(input.shape());
  size_t valSize = GetTotalSize(val_.shape());

  UTIL_THROW_IF2(inputSize != GetTotalSize(grad.shape()),
			  "inputSize != gradSize:" << inputSize << "!=" << GetTotalSize(grad.shape()));
  UTIL_THROW_IF2(valSize != GetTotalSize(adj_.shape()),
			  "valSize != adjSize :" << valSize << "!=" << GetTotalSize(adj_.shape()));

  cerr	<< "inputSize=grad=" << Debug(input.shape())<< "=" << inputSize << " "
		<< "valSize=adj_=" << Debug(val_.shape()) << "=" << valSize
		<< endl;

  //cerr << "input=" << input.Debug() << endl;
  //cerr << "adj_=" << adj_.Debug() << endl;

  std::vector<float> prevCalcGrad;
  prevCalcGrad << grad;
  //cerr << "origGrad=" << grad.Debug() << endl;
  //output("diffGrad", diffGrad);

  //output("prevCalcGrad", prevCalcGrad.begin(), prevCalcGrad.end());

  Tensor newValTensor(input.shape());

  // LOOP thru each element in input & add delta
  for (size_t inputInd = 0; inputInd < inputSize; ++inputInd) {
	  input.incr(inputInd, delta);
	  //output("input", input.begin(), input.end());

	  forward();

	  val_.sum(newValTensor, inputInd);
	  //hipDeviceSynchronize();

	  input.incr(inputInd, -delta);
  }

  std::vector<float> newVal;
  newVal << newValTensor;
  //hipDeviceSynchronize();

  // orig value
  forward();

  float sumValOrig = val_.sum();
  //float sumValOrig = thrust::reduce(val_.begin(), val_.end(), (float) 0.0f, thrust::plus<float>());
  //hipDeviceSynchronize();

  //output("newVal", newVal.begin(), newVal.end());

  // calc gradient
  Tensor prevGradTensor(input.shape());
  thrust::copy(grad.begin(), grad.end(), prevGradTensor.begin());

  Tensor gradTensor(input.shape());
  Element(_1 = (_2 - sumValOrig) / delta, gradTensor, newValTensor);
  Element(_1 = _2 * _3 + _4, grad, adj_, gradTensor, prevGradTensor);
}

void Node::outputL2Norm(const std::string &str, const std::vector<float> &x, const std::vector<float> &y) const
{
  using namespace std;
  // print out diff between diffGradA and numericalGrad
  if(x.size() != y.size()) {
	cerr << "size error: " << x.size() << "!=" << y.size() << endl;
	exit(1);
  }

  std::vector<float> diff(x.size());
  for (size_t i = 0; i < x.size(); ++i) {
	  diff[i] = x[i] - y[i];
  }
  cerr << "L2-norm of difference " << typeid(*this).name() << ":" << str << "=" << L2Norm(diff) << endl << endl;
}

float Node::L2Norm(const std::vector<float> &vec) const
{
  float ret = 0;
  for (size_t i = 0; i < vec.size(); ++i) {
	  ret += vec[i] * vec[i];
  }
  return sqrt(ret);
}

}

