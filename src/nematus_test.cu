#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <iomanip>
#include <string>
#include <cstdio>
#include <boost/timer/timer.hpp>

#include "marian.h"
#include "nematus.h"
#include "batch_generator.h"
#include "optimizers.h"

#include "corpus.h"

int main(int argc, char** argv) {
  using namespace marian;
  using namespace data;

  hipSetDevice(0);

  std::vector<std::string> files =
    {"/work/wmt16/work/unbabel/wmt2015/APE/train.mt-pe.gpu0/train.all.mt",
     "/work/wmt16/work/unbabel/wmt2015/APE/train.mt-pe.gpu0/train.all.pe"};

  std::vector<std::string> vocab =
    {"/work/wmt16/work/unbabel/wmt2015/APE/train.mt-pe.gpu0/mt.json",
     "/work/wmt16/work/unbabel/wmt2015/APE/train.mt-pe.gpu0/pe.json"};

  auto corpus = DataSet<Corpus>(files, vocab, 50);
  BatchGenerator<Corpus> bg(corpus, 40, 1000);

  auto nematus = New<Nematus>();
  nematus->reserveWorkspaceMB(6144);
  auto opt = Optimizer<Adam>(0.0001
                             /*,clip=norm(1)*/);

  float sum = 0;
  boost::timer::cpu_timer timer;
  size_t batches = 1;
  for(int i = 1; i <= 20; ++i) {
    bg.prepare();
    while(bg) {
      auto batch = bg.next();

      nematus->construct(*batch);
      opt->update(nematus);

      float cost = nematus->cost();
      sum += cost;

      if(batches % 1 == 0)
        std::cerr << ".";
      if(batches % 100 == 0)
        std::cout << "[" << batches << "]" << std::fixed << std::setfill(' ') << std::setw(9)
                  << " - cost: " << cost << "/" << sum / batches
                  << " - time: " << timer.format(5, "%ws") << std::endl;

      if(batches % 10000 == 0)
        nematus->save("../test/model.marian.npz");

      batches++;
    }
  }
  std::cout << std::endl;
  std::cout << timer.format(5, "%ws") << std::endl;

  return 0;
}
