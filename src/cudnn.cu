#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hipDNN.h>

#include <boost/timer/timer.hpp>

#include "tensor.h"
#include "tensor_operators.h"
#include "param_initializers.h"

using namespace marian;

void CudnnSoftmaxForward(hipdnnHandle_t cudnnHandle,
                  Tensor out, Tensor in) {
    float alpha = 1, beta = 0;
    hipdnnSoftmaxForward(cudnnHandle,
                        HIPDNN_SOFTMAX_LOG,
                        HIPDNN_SOFTMAX_MODE_CHANNEL,
                        &alpha,
                        in.cudnn(),
                        in.data(),
                        &beta,
                        out.cudnn(),
                        out.data());
    hipDeviceSynchronize();
}

void CudnnSoftmaxBackward(hipdnnHandle_t cudnnHandle,
                          Tensor out, Tensor in) {
    float alpha = 1, beta = 0;
    hipdnnSoftmaxBackward(cudnnHandle,
                         HIPDNN_SOFTMAX_LOG,
                         HIPDNN_SOFTMAX_MODE_CHANNEL,
                         &alpha,
                         in.cudnn(),
                         in.data(),
                         out.cudnn(),
                         out.data(),
                         &beta,
                         out.cudnn(),
                         out.data());
    hipDeviceSynchronize();
}

int main() {
    hipdnnHandle_t cudnnHandle;
    hipdnnCreate(&cudnnHandle);
    
    int d = 10;
    
    Tensor in({d, d});
    Tensor out({d, d});
    Tensor grad({d, d});
    Tensor adj({d, d}, 1);
    
    auto f = uniform(-5, 5);
    f(in);
    
    std::cerr << in.Debug() << std::endl;
    
    {
        boost::timer::cpu_timer timer;
        for(int i = 0; i < 1; ++i) {
          CudnnSoftmaxForward(cudnnHandle, out, in);
          std::cerr << out.Debug() << std::endl;
          CudnnSoftmaxBackward(cudnnHandle, grad, in);
          std::cerr << grad.Debug() << std::endl;
        }
      
        std::cerr << timer.format(5, "%ws") << std::endl;
    }
    
    {
        boost::timer::cpu_timer timer;
        for(int i = 0; i < 1; ++i) {
          Element(_1 = _2, out, in);
          Softmax(&out);
          std::cerr << out.Debug() << std::endl;
          SoftmaxGrad(grad, adj, out);
          std::cerr << grad.Debug() << std::endl; 
        }
        //std::cerr << grad.Debug() << std::endl;
        std::cerr << timer.format(5, "%ws") << std::endl;
    }
    
    
    //// Copy back
    //float *result = (float *) malloc(m * c * sizeof(float));
    //hipMemcpy(result, d_softmaxData, m * c * sizeof(float), hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();
    //
    //// Log
    //printf("SOFTMAX:\n");
    //printMatrix(result, c, m);
    //
    //// Try backward
    //hipdnnTensorDescriptor_t diffTensorDesc;
    //hipdnnCreateTensorDescriptor(&diffTensorDesc);
    //hipdnnSetTensor4dDescriptor(diffTensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
    //                           m, c, 1, 1);
    //
    //float *d_gradData;
    //hipMalloc((void**) &d_gradData, m * c * sizeof(float));
    //
    //float *diffData = makeDiffData(m, c);
    //float *d_diffData;
    //hipMalloc((void**) &d_diffData, m * c * sizeof(float));
    //hipMemcpy(d_diffData, diffData, m * c * sizeof(float), hipMemcpyHostToDevice);
    //hipDeviceSynchronize();
    //
    //hipdnnSoftmaxBackward(cudnnHandle,
    //                     HIPDNN_SOFTMAX_ACCURATE,
    //                     HIPDNN_SOFTMAX_MODE_CHANNEL,
    //                     &alpha,
    //                     srcTensorDesc,
    //                     d_softmaxData,
    //                     diffTensorDesc,
    //                     d_diffData,
    //                     &beta,
    //                     sftTensorDesc,
    //                     d_gradData);
    //hipDeviceSynchronize();
    //
    //// Copy back
    //float *result_backward = (float *) malloc(m * c * sizeof(float));
    //hipMemcpy(result_backward, d_gradData, m * c * sizeof(float), hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();
    //
    //// Log
    //printf("GRADIENT:\n");
    //printMatrix(result_backward, c, m);
    //
    //// Destruct
    //free(result);
    //free(diffData);
    //free(result_backward);
    //free(fcLayer);
    //
    //hipdnnDestroyTensorDescriptor(srcTensorDesc);
    //hipdnnDestroyTensorDescriptor(sftTensorDesc);
    //hipdnnDestroyTensorDescriptor(diffTensorDesc);
    //hipFree(d_fcLayer);
    //hipFree(d_softmaxData);
    //hipFree(d_gradData);
    //hipFree(d_diffData);
    hipdnnDestroy(cudnnHandle);
    return 0;
}