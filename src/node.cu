#include "hip/hip_runtime.h"
#include "node.h"
#include "tensor_operators.h"

namespace marian {

// for backward_numeric
void Node::calc_numeric_grad(
		  Float delta,
		  Tensor input,
		  Tensor grad
		  )
{
  using namespace std;

  size_t inputSize = GetTotalSize(input.shape());
  size_t valSize = GetTotalSize(val_.shape());

  UTIL_THROW_IF2(inputSize != GetTotalSize(grad.shape()),
			  "inputSize != gradSize:" << inputSize << "!=" << GetTotalSize(grad.shape()));
  UTIL_THROW_IF2(valSize != GetTotalSize(adj_.shape()),
			  "valSize != adjSize :" << valSize << "!=" << GetTotalSize(adj_.shape()));

  cerr	<< "inputSize=grad=" << Debug(input.shape())<< "=" << inputSize << " "
		<< "valSize=adj_=" << Debug(val_.shape()) << "=" << valSize
		<< endl;

  //cerr << "input=" << input.Debug() << endl;
  //cerr << "adj_=" << adj_.Debug() << endl;

  std::vector<float> prevCalcGrad;
  prevCalcGrad << grad;
  //cerr << "origGrad=" << grad.Debug() << endl;
  //output("diffGrad", diffGrad);

  //output("prevCalcGrad", prevCalcGrad.begin(), prevCalcGrad.end());

  Tensor newValTensor(input.shape());

  // LOOP thru each element in input & add delta
  for (size_t inputInd = 0; inputInd < inputSize; ++inputInd) {
	  input.incr(inputInd, delta);
	  //output("input", input.begin(), input.end());

	  forward();

	  val_.sum(newValTensor, inputInd);

	  input.incr(inputInd, -delta);
  }

  std::vector<float> newVal;
  newVal << newValTensor;
  hipDeviceSynchronize();

  // orig value
  forward();

  float sumValOrig = val_.sum();

  //output("newVal", newVal.begin(), newVal.end());

  // calc gradient
  Tensor prevGradTensor(input.shape());
  thrust::copy(grad.begin(), grad.end(), prevGradTensor.begin());

  Tensor gradTensor(input.shape());
  Element(_1 = (_2 - sumValOrig) / delta, gradTensor, newValTensor);
  Element(_1 = _2 * _3 + _4, grad, adj_, gradTensor, prevGradTensor);
}

void Node::broadcast(const std::vector<float> &largeVec, std::vector<float> &smallVec)
{
	size_t largeSize = largeVec.size();
	size_t smallSize = smallVec.size();

    UTIL_THROW_IF2(largeSize < smallSize,
    		"largeSize < smallSize:" << largeSize << "<" << smallSize);
    UTIL_THROW_IF2(largeSize % smallSize,
    		"largeSize % smallSize != 0:" << largeSize << " " << smallSize);

    smallVec.resize(largeSize);
    for (size_t i = smallSize; i < largeSize; i += smallSize) {
    	std::copy(smallVec.begin(), smallVec.begin() + smallSize, smallVec.begin() + i);
    }
}

void Node::outputL2Norm(const std::string &str, const std::vector<float> &x, const std::vector<float> &y) const
{
  using namespace std;
  // print out diff between diffGradA and numericalGrad
  if(x.size() != y.size()) {
	cerr << "size error: " << x.size() << "!=" << y.size() << endl;
	exit(1);
  }

  std::vector<float> diff(x.size());
  for (size_t i = 0; i < x.size(); ++i) {
	  diff[i] = x[i] - y[i];
  }
  cerr << "L2-norm of difference " << typeid(*this).name() << ":" << str << "=" << L2Norm(diff) << endl << endl;
}

float Node::L2Norm(const std::vector<float> &vec) const
{
  float ret = 0;
  for (size_t i = 0; i < vec.size(); ++i) {
	  ret += vec[i] * vec[i];
  }
  return sqrt(ret);
}

}

