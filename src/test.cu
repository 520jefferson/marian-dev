#include "hip/hip_runtime.h"
#include <fstream>
#include "marian.h"
#include "mnist.h"
#include "vocab.h"

#include "tensor_operators.h"

using namespace std;

///////////////////////////////////////////////////////
__global__ void gArgMax(float* arr, size_t rows, size_t cols) {
  for (size_t row = 0; row < rows; ++row) {
    size_t startInd = row * cols;
    float maxScore = -99999;
    size_t maxInd = -1;
    for (size_t col = 0; col < cols; ++col) {
      size_t ind = startInd + col;
      float score = arr[ind];
      if (score > maxScore) {
        maxScore = score;
        maxInd = col;
      }
    }
    arr[startInd] = maxInd;
  }
}

__global__ void gArgMax2(float* arr, size_t rows, size_t cols) {
	size_t row = blockIdx.x;
    size_t startInd = row * cols;
    float maxScore = -99999;
    size_t maxInd = -1;
    for (size_t col = 0; col < cols; ++col) {
      size_t ind = startInd + col;
      float score = arr[ind];
      if (score > maxScore) {
        maxScore = score;
        maxInd = col;
      }
    }
    arr[startInd] = maxInd;
}

string output(const std::vector<float> &vec)
{
  stringstream strm;
  for (size_t i = 0; i < vec.size(); ++i) {
  strm << vec[i] << " ";
  }
  return strm.str();
}

void temp()
{
  using namespace std;
  using namespace marian;

	std::vector<float> hVec({29,19,  49,39,  79,99,  79,39});
        cerr << "hVec =" << output(hVec) << endl;

	thrust::device_vector<float> dVec(8);
	thrust::copy(hVec.begin(), hVec.end(), dVec.begin());
	float *data = thrust::raw_pointer_cast(dVec.data());

	//gArgMax<<<10, 20, sizeof(float)>>>(data, 4, 2);
	gArgMax2<<<4, 1, sizeof(float)>>>(data, 4, 2);

	std::vector<float> hVec2(8);
	thrust::copy(dVec.begin(), dVec.end(), hVec2.begin());
	cerr << "hVec2=" << output(hVec2) << endl;

	exit(0);
}

///////////////////////////////////////////////////////
int main(int argc, char** argv) {
  temp();

  using namespace std;
  using namespace marian;
  using namespace keywords;

  Vocab sourceVocab, targetVocab;

  int input_size = 10;
  int output_size = 2;
  int batch_size = 25;
  int hidden_size = 5;
  int num_inputs = 8;

  std::vector<Expr> X;
  std::vector<Expr> Y;
  std::vector<Expr> H;

  ExpressionGraph g(0);

  for (int t = 0; t < num_inputs; ++t) {
    X.emplace_back(g.input(shape={batch_size, input_size}));
    Y.emplace_back(g.input(shape={batch_size, output_size}));
  }

  Expr Wxh = g.param(shape={input_size, hidden_size}, init=uniform(), name="Wxh");
  Expr Whh = g.param(shape={hidden_size, hidden_size}, init=uniform(), name="Whh");
  Expr bh = g.param(shape={1, hidden_size}, init=uniform(), name="bh");
  Expr h0 = g.param(shape={1, hidden_size}, init=uniform(), name="h0");

  // read parallel corpus from file
  std::fstream sourceFile("../examples/mt/dev/newstest2013.de");
  std::fstream targetFile("../examples/mt/dev/newstest2013.en");

  string sourceLine, targetLine;
  while (getline(sourceFile, sourceLine)) {
	  getline(targetFile, targetLine);

	  std::vector<size_t> sourceIds = sourceVocab.ProcessSentence(sourceLine);
	  std::vector<size_t> targetIds = sourceVocab.ProcessSentence(targetLine);
  }

  std::cerr << "Building RNN..." << std::endl;
  H.emplace_back(tanh(dot(X[0], Wxh) + dot(h0, Whh) + bh));
  for (int t = 1; t < num_inputs; ++t) {
    H.emplace_back(tanh(dot(X[t], Wxh) + dot(H[t-1], Whh) + bh));
  }

  Expr Why = g.param(shape={hidden_size, output_size}, init=uniform(), name="Why");
  Expr by = g.param(shape={1, output_size}, init=uniform(), name="by");

  std::cerr << "Building output layer..." << std::endl;
  std::vector<Expr> Yp;

  Yp.emplace_back(softmax_fast(dot(H[0], Why) + by));
  Expr cross_entropy = sum(Y[0] * log(Yp[0]), axis=1);
  for (int t = 1; t < num_inputs; ++t) {
    Yp.emplace_back(softmax_fast(dot(H[t], Why) + by));
    cross_entropy = cross_entropy + sum(Y[t] * log(Yp[t]), axis=1);
  }
  auto graph = -mean(cross_entropy, axis=0, name="cost");

  for (int t = 0; t < num_inputs; ++t) {
    Tensor Xt({batch_size, input_size});
    Tensor Yt({batch_size, output_size});

    float max = 1.;
    std::vector<float> values(batch_size * input_size);
    std::vector<float> classes(batch_size * output_size, 0.0);
    int k = 0;
    int l = 0;
    for (int i = 0; i < batch_size; ++i) {
      for (int j = 0; j < input_size; ++j, ++k) {
         values[k] = max * (2.0*static_cast<float>(rand()) / RAND_MAX - 1.0);
      }
      int gold = output_size * static_cast<float>(rand()) / RAND_MAX;
      classes[l + gold] = 1.0;
      l += output_size;
    }

    thrust::copy(values.begin(), values.end(), Xt.begin());
    thrust::copy(classes.begin(), classes.end(), Yt.begin());

    X[t] = Xt;
    Y[t] = Yt;
  }

  std::cout << g.graphviz() << std::endl;
  
  g.forward(batch_size);
  g.backward();

  std::cerr << graph.val().Debug() << std::endl;

  std::cerr << X[0].val().Debug() << std::endl;
  std::cerr << Y[0].val().Debug() << std::endl;

  std::cerr << Whh.grad().Debug() << std::endl;
  std::cerr << bh.grad().Debug() << std::endl;
  std::cerr << Why.grad().Debug() << std::endl;
  std::cerr << by.grad().Debug() << std::endl;
  std::cerr << Wxh.grad().Debug() << std::endl;
  std::cerr << h0.grad().Debug() << std::endl;

  return 0;
}
