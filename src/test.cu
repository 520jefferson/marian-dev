#include "hip/hip_runtime.h"

#include "marian.h"

using namespace std;

int main(int argc, char** argv) {

  using namespace marian;
  using namespace keywords;
  
  /*
  Expr x = input(shape={whatevs, 784}, name="X");
  Expr y = input(shape={whatevs, 10}, name="Y");
  
  Expr w = param(shape={784, 10}, name="W0");
  Expr b = param(shape={1, 10}, name="b0");
  
  Expr n5 = dot(x, w);
  Expr n6 = n5 + b;
  Expr lr = softmax(n6, axis=1, name="pred");
  cerr << "lr=" << lr.Debug() << endl;

  Expr graph = -mean(sum(y * log(lr), axis=1), axis=0, name="cost");
  
  Tensor tx({500, 784}, 1);
  Tensor ty({500, 10}, 1);
  cerr << "tx=" << tx.Debug() << endl;
  cerr << "ty=" << ty.Debug() << endl;

  x = tx;
  y = ty;

  graph.forward(500);
  //std::cerr << graph["pred"].val()[0] << std::endl;
  
  */

  Expr x = input(shape={whatevs, 2}, name="X");
  Expr y = input(shape={whatevs, 2}, name="Y");

  Expr w = param(shape={2, 1}, name="W0");
  Expr b = param(shape={1, 1}, name="b0");

  Expr n5 = dot(x, w);
  Expr n6 = n5 + b;
  Expr lr = softmax(n6, axis=1, name="pred");
  cerr << "lr=" << lr.Debug() << endl;

  Expr graph = -mean(sum(y * log(lr), axis=1), axis=0, name="cost");

  Tensor tx({4, 2}, 1);
  Tensor ty({4, 1}, 1);
  cerr << "tx=" << tx.Debug() << endl;
  cerr << "ty=" << ty.Debug() << endl;

  tx.Load("../examples/xor/train.txt");
  ty.Load("../examples/xor/label.txt");


  //hook0(graph);
  //graph.autodiff();
  //std::cerr << graph["cost"].val()[0] << std::endl;
  //hook1(graph);
  //for(auto p : graph.params()) {
  //  auto update = _1 = _1 - alpha * _2;
  //  Element(update, p.val(), p.grad());
  //}
  //hook2(graph);
  //
  //auto opt = adadelta(cost_function=cost,
  //                    eta=0.9, gamma=0.1,
  //                    set_batch=set,
  //                    before_update=before,
  //                    after_update=after,
  //                    set_valid=valid,
  //                    validation_freq=100,
  //                    verbose=1, epochs=3, early_stopping=10);
  //opt.run();

  return 0;
}
