#include "tensor_operators.h"

namespace marian {

Tensor Prod(hipblasHandle_t handle, Tensor C, const Tensor A, const Tensor B,
             bool transA, bool transB, Float beta) {
  Float alpha = 1.0;

  size_t m = A.shape()[0];
  size_t k = A.shape()[1];
  if(transA)
    std::swap(m, k);
  
  size_t l = B.shape()[0];
  size_t n = B.shape()[1];
  if(transB)
    std::swap(l, n);
  
  size_t lda = A.shape()[1];
  size_t ldb = B.shape()[1];
  size_t ldc = B.shape()[1];
  
  if(transB)
    ldc = B.shape()[0];
  
  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  
  hipblasSgemm(handle, opB, opA,
              n, m, k, &alpha, B.data(), ldb, A.data(), lda, &beta, C.data(), ldc);
  return C;
}

Tensor Prod(Tensor C, const Tensor A, const Tensor B,
             bool transA, bool transB, Float beta) {

  return Prod(handles.cublasHandle, C, A, B, transA, transB, beta);
}

}