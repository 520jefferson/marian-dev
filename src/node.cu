#include "hip/hip_runtime.h"
#include "node.h"

namespace marian {

// for backward_numeric
void Node::calc_numeric_grad(
		  Float delta,
		  Tensor input,
		  Tensor grad
		  )
{
  using namespace std;

  size_t inputSize = GetTotalSize(input.shape());
  size_t valSize = GetTotalSize(val_.shape());

  UTIL_THROW_IF2(inputSize != GetTotalSize(grad.shape()),
			  "inputSize != gradSize:" << inputSize << "!=" << GetTotalSize(grad.shape()));
  UTIL_THROW_IF2(valSize != GetTotalSize(adj_.shape()),
			  "valSize != adjSize :" << valSize << "!=" << GetTotalSize(adj_.shape()));

  cerr	<< "inputSize=grad=" << Debug(input.shape())<< "=" << inputSize << " "
		<< "valSize=adj_=" << Debug(val_.shape()) << "=" << valSize
		<< endl;

  //cerr << "input=" << input.Debug() << endl;
  //cerr << "adj_=" << adj_.Debug() << endl;

  std::vector<float> prevCalcGrad;
  prevCalcGrad << grad;
  //cerr << "origGrad=" << grad.Debug() << endl;
  //output("diffGrad", diffGrad);

  //output("prevCalcGrad", prevCalcGrad.begin(), prevCalcGrad.end());

  std::vector<float> inputVec;
  inputVec << input;
  //output("inputVec", inputVec);

  Tensor newValTensor(input.shape());

  // LOOP thru each element in input & add delta
  for (size_t inputInd = 0; inputInd < inputSize; ++inputInd) {
	  inputVec[inputInd] += delta;
	  input << inputVec;
	  //output("input", input.begin(), input.end());

	  forward();

	  val_.sum(newValTensor, inputInd);

	  inputVec[inputInd] -= delta;
  }

  std::vector<float> newVal;
  newVal << newValTensor;
  hipDeviceSynchronize();

  // orig value
  input << inputVec;
  forward();

  float sumValOrig = val_.sum();

  //output("newVal", newVal.begin(), newVal.end());

  // calc gradient
  //cerr << "adj_=" << adj_.Debug() << endl;
  std::vector<float> adjVec;
  adjVec << adj_;

  std::vector<float> numericalGrad(inputSize);
  for (size_t i = 0; i < numericalGrad.size(); ++i) {
	  numericalGrad[i] = (newVal[i] - sumValOrig) / delta;
  }

  broadcast(numericalGrad, adjVec);
  //std::cerr << "broadcast size=" << numericalGrad.size() << " " << adjVec.size() << std::endl;
  //output("adjVec=", adjVec.begin(), adjVec.end());

  for (size_t i = 0; i < numericalGrad.size(); ++i) {
	  numericalGrad[i] *= adjVec[i];
	  numericalGrad[i] += prevCalcGrad[i];
  }

  //output("prevCalcGrad=", prevCalcGrad.begin(), prevCalcGrad.end());
  //output("adjVec=", adjVec.begin(), adjVec.end());

  // set grad results
  grad << numericalGrad;
  //output("numericalGrad", numericalGrad);
}

void Node::broadcast(const std::vector<float> &largeVec, std::vector<float> &smallVec)
{
	size_t largeSize = largeVec.size();
	size_t smallSize = smallVec.size();

    UTIL_THROW_IF2(largeSize < smallSize,
    		"largeSize < smallSize:" << largeSize << "<" << smallSize);
    UTIL_THROW_IF2(largeSize % smallSize,
    		"largeSize % smallSize != 0:" << largeSize << " " << smallSize);

    smallVec.resize(largeSize);
    for (size_t i = smallSize; i < largeSize; i += smallSize) {
    	std::copy(smallVec.begin(), smallVec.begin() + smallSize, smallVec.begin() + i);
    }
}

void Node::outputL2Norm(const std::string &str, const std::vector<float> &x, const std::vector<float> &y) const
{
  using namespace std;
  // print out diff between diffGradA and numericalGrad
  if(x.size() != y.size()) {
	cerr << "size error: " << x.size() << "!=" << y.size() << endl;
	exit(1);
  }

  std::vector<float> diff(x.size());
  for (size_t i = 0; i < x.size(); ++i) {
	  diff[i] = x[i] - y[i];
  }
  cerr << "L2-norm of difference " << typeid(*this).name() << ":" << str << "=" << L2Norm(diff) << endl << endl;
}

float Node::L2Norm(const std::vector<float> &vec) const
{
  float ret = 0;
  for (size_t i = 0; i < vec.size(); ++i) {
	  ret += vec[i] * vec[i];
  }
  return sqrt(ret);
}

}

