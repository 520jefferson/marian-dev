#include "dropout_cudnn.h"

#include "tensors/tensor.h"

namespace marian {

static hipdnnHandle_t create_handle_dnn() {
 hipdnnHandle_t cudnnHandle;
 hipdnnCreate(&cudnnHandle);
 return cudnnHandle;
}

hipdnnHandle_t cudnnHandle = create_handle_dnn();

void CudnnDropoutPrepare(Tensor in, float p,
                        hipdnnDropoutDescriptor_t* dropDesc,
                        void** space, size_t* spaceSize,
                        void** states, size_t seed) {
 size_t statesSize;
 hipdnnDropoutGetStatesSize(cudnnHandle, &statesSize);
 cudnnDropoutGetReserveSpaceSize(in->cudnn(), spaceSize);

 hipMalloc((void**)states, statesSize);
 hipMalloc((void**)space, *spaceSize);

 hipdnnCreateDropoutDescriptor(dropDesc);
 hipdnnSetDropoutDescriptor(*dropDesc,
                           cudnnHandle,
                           p,
                           (void*)*states,
                           statesSize,
                           seed);
}

void CudnnDropoutDestroy(hipdnnDropoutDescriptor_t dropDesc,
                        void* space, void* states) {
 hipdnnDestroyDropoutDescriptor(dropDesc);
 hipFree(space);
 hipFree(states);
}

void CudnnDropoutForward(hipdnnDropoutDescriptor_t dropoutDesc,
                 void* space, size_t spaceSize,
                 Tensor out, Tensor in) {
 cudnnDropoutForward(cudnnHandle,
                     dropoutDesc,
                     in->cudnn(),
                     in->data(),
                     out->cudnn(),
                     out->data(),
                     space,
                     spaceSize);
}

/* void CudnnDropoutBackward(hipdnnDropoutDescriptor_t dropoutDesc, */
                         /* void* space, size_t spaceSize, */
                         /* Tensor out, Tensor in) { */
 /* auto inGpu = static_cast<TensorGPU*>(in.get()); */
 /* auto outGpu = static_cast<TensorGPU*>(out.get()); */
 /* cudnnDropoutBackward(cudnnHandle, */
                     /* dropoutDesc, */
                     /* inGpu->cudnn(), */
                     /* inGpu->data(), */
                     /* outGpu->cudnn(), */
                     /* outGpu->data(), */
                     /* space, */
                     /* spaceSize); */
/* } */

}
