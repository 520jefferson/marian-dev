#include <iostream>
#include <boost/timer/timer.hpp>

//#include "tensors/tensor_allocator.h"
//#include "tensors/tensor_gpu.h"
//#include "kernels/tensor_operators.h"
//#include "kernels/thrust_functions.h"

#include "data/corpus.h"
#include "data/batch_generator.h"

using namespace marian;



int main() {

  std::vector<std::string> files =
    {"../benchmark/train.tok.true.en",
     "../benchmark/train.tok.true.en",
     "../benchmark/train.tok.true.de"};

  std::vector<std::string> vocab =
    {"../benchmark/train.tok.true.en.json",
     "../benchmark/train.tok.true.en.json",
     "../benchmark/train.tok.true.de.json"};

  std::vector<int> maxVocab = { 50000, 50000, 50000 };

  using namespace data;
  auto corpus = New<Corpus>(files, vocab, maxVocab, 50);
  BatchGenerator<Corpus> bg(corpus, 64, 20);

  bg.prepare();

  size_t i = 0;
  size_t samples = 0;
  while(bg) {
    auto batch = bg.next();
    if(i && i % 10000 == 0)
      std::cerr << "[" << i << "/" << samples << "]" << std::endl;
    if(i % 100 == 0)
      std::cerr << ".";
    i++;

    samples += batch->size();
  }




  //TensorAllocator params = newTensorAllocator<DeviceGPU>();
  //
  //hipblasHandle_t handle = create_handle();
  //
  //int words = 64;
  //int batch = 128;
  //int hidden = 4096;
  //
  //Tensor mappedState;
  //params->allocate(mappedState, {batch, hidden, 1});
  //mappedState->set(0.001);
  //
  //Tensor mappedContext;
  //params->allocate(mappedContext, {batch, hidden, words});
  //mappedContext->set(0.001);
  //
  //Tensor va;
  //params->allocate(va, {hidden, 1});
  //va->set(0.001);
  //
  //Tensor out1;
  //params->allocate(out1, {batch, hidden, words});
  //out1->set(0);
  //
  //Tensor gMappedState;
  //params->allocate(gMappedState, {batch, hidden, 1});
  //gMappedState->set(0);
  //
  //Tensor gMappedContext;
  //params->allocate(gMappedContext, {batch, hidden, words});
  //gMappedContext->set(0.001);
  //
  //Tensor gVa;
  //params->allocate(gVa, {hidden, 1});
  //va->set(0.001);
  //
  //Tensor gOut1;
  //params->allocate(gOut1, {batch, hidden, words});
  //out1->set(0);
  //
  //Tensor out2;
  //params->allocate(out2, {batch, 1, words});
  //out2->set(0);
  //
  //boost::timer::cpu_timer timer;
  //for(int i = 0; i < 5000; ++i) {
  //  Element(_1 = Tanh(_2 + _3), out1, mappedState, mappedContext);
  //  Prod(handle, out2, out1, va, false, false, 0);
  //  Prod(handle, gOut1, out2, va, false, true, 1.0f);
  //  Prod(handle, gVa, out1, out2, true, false, 1.0f);
  //  Add(_1 * (1.f - (_2 *_2)), gMappedState, out1, out1);
  //  Add(_1 * (1.f - (_2 *_2)), gMappedContext, out1, out1);
  //  hipStreamSynchronize(0);
  //
  //  if(i % 100 == 0)
  //    std::cout << "." << std::flush;
  //}
  //std::cout << timer.format(5, "%ws") << std::endl;
  //
  //boost::timer::cpu_timer timer2;
  //for(int i = 0; i < 5000; ++i) {
  //  Att(out2, mappedContext, mappedState, va);
  //  AttBack(gMappedContext, gMappedState, gVa,
  //      mappedContext, mappedState, va, out2);
  //  hipStreamSynchronize(0);
  //  if(i % 100 == 0)
  //    std::cout << "." << std::flush;
  //}
  //std::cout << timer2.format(5, "%ws") << std::endl;

  return 0;
}
