#include "hip/hip_runtime.h"
#include <fstream>
#include "marian.h"
#include "mnist.h"
#include "vocab.h"

int main(int argc, char** argv) {
  hipSetDevice(0);

  using namespace std;
  using namespace marian;
  using namespace keywords;

  Vocab sourceVocab, targetVocab;

  int input_size = 10;
  int output_size = 2;
  int batch_size = 25;
  int hidden_size = 5;
  int num_inputs = 8;

  std::vector<Expr> X;
  std::vector<Expr> Y;
  std::vector<Expr> H;

  ExpressionGraph g;

  for (int t = 0; t < num_inputs; ++t) {
    X.emplace_back(g.input(shape={batch_size, input_size}));
    Y.emplace_back(g.input(shape={batch_size, output_size}));
  }

  Expr Wxh = g.param(shape={input_size, hidden_size}, init=uniform(), name="Wxh");
  Expr Whh = g.param(shape={hidden_size, hidden_size}, init=uniform(), name="Whh");
  Expr bh = g.param(shape={1, hidden_size}, init=uniform(), name="bh");
  Expr h0 = g.param(shape={1, hidden_size}, init=uniform(), name="h0");

  // read parallel corpus from file
  std::fstream sourceFile("../examples/mt/dev/newstest2013.de");
  std::fstream targetFile("../examples/mt/dev/newstest2013.en");

  string sourceLine, targetLine;
  while (getline(sourceFile, sourceLine)) {
	  getline(targetFile, targetLine);

	  std::vector<size_t> sourceIds = sourceVocab.ProcessSentence(sourceLine);
	  std::vector<size_t> targetIds = sourceVocab.ProcessSentence(targetLine);
  }

  std::cerr << "Building RNN..." << std::endl;
  H.emplace_back(tanh(dot(X[0], Wxh) + dot(h0, Whh) + bh));
  for (int t = 1; t < num_inputs; ++t) {
    H.emplace_back(tanh(dot(X[t], Wxh) + dot(H[t-1], Whh) + bh));
  }

  Expr Why = g.param(shape={hidden_size, output_size}, init=uniform(), name="Why");
  Expr by = g.param(shape={1, output_size}, init=uniform(), name="by");

  std::cerr << "Building output layer..." << std::endl;
  std::vector<Expr> Yp;

  Yp.emplace_back(softmax_fast(dot(H[0], Why) + by));
  Expr cross_entropy = sum(Y[0] * log(Yp[0]), axis=1);
  for (int t = 1; t < num_inputs; ++t) {
    Yp.emplace_back(softmax_fast(dot(H[t], Why) + by));
    cross_entropy = cross_entropy + sum(Y[t] * log(Yp[t]), axis=1);
  }
  auto graph = -mean(cross_entropy, axis=0, name="cost");

  for (int t = 0; t < num_inputs; ++t) {
    Tensor Xt({batch_size, input_size});
    Tensor Yt({batch_size, output_size});

    float max = 1.;
    std::vector<float> values(batch_size * input_size);
    std::vector<float> classes(batch_size * output_size, 0.0);
    int k = 0;
    int l = 0;
    for (int i = 0; i < batch_size; ++i) {
      for (int j = 0; j < input_size; ++j, ++k) {
         values[k] = max * (2.0*static_cast<float>(rand()) / RAND_MAX - 1.0);
      }
      int gold = output_size * static_cast<float>(rand()) / RAND_MAX;
      classes[l + gold] = 1.0;
      l += output_size;
    }

    thrust::copy(values.begin(), values.end(), Xt.begin());
    thrust::copy(classes.begin(), classes.end(), Yt.begin());

    X[t] = Xt;
    Y[t] = Yt;
  }

  std::cout << g.graphviz() << std::endl;
  
  g.forward(batch_size);
  g.backward();

  std::cerr << graph.val().Debug() << std::endl;

  std::cerr << X[0].val().Debug() << std::endl;
  std::cerr << Y[0].val().Debug() << std::endl;

  std::cerr << Whh.grad().Debug() << std::endl;
  std::cerr << bh.grad().Debug() << std::endl;
  std::cerr << Why.grad().Debug() << std::endl;
  std::cerr << by.grad().Debug() << std::endl;
  std::cerr << Wxh.grad().Debug() << std::endl;
  std::cerr << h0.grad().Debug() << std::endl;

  return 0;
}
