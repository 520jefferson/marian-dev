#include "hip/hip_runtime.h"

#include "marian.h"
#include "mnist.h"

using namespace std;

int main(int argc, char** argv) {
  /*int numImg = 0;*/
  /*auto images = datasets::mnist::ReadImages("../examples/mnist/t10k-images-idx3-ubyte", numImg);*/
  /*auto labels = datasets::mnist::ReadLabels("../examples/mnist/t10k-labels-idx1-ubyte", numImg);*/

  using namespace marian;
  using namespace keywords;
  
  const size_t BATCH_SIZE = 500;
  const size_t IMAGE_SIZE = 784;
  const size_t LABEL_SIZE = 10;

  Expr x = input(shape={whatevs, IMAGE_SIZE}, name="X");
  Expr y = input(shape={whatevs, LABEL_SIZE}, name="Y");
  
  Expr w = param(shape={IMAGE_SIZE, LABEL_SIZE}, name="W0");
  Expr b = param(shape={1, LABEL_SIZE}, name="b0");
    
  Expr z = dot(x, w) + b;
  Expr lr = softmax(z, axis=1, name="pred");
  Expr graph = -mean(sum(y * log(lr), axis=1), axis=0, name="cost");
  //cerr << "x=" << Debug(lr.val().shape()) << endl;

  int numofdata;
  //vector<float> images = datasets::mnist::ReadImages("../examples/mnist/t10k-images-idx3-ubyte", numofdata, IMAGE_SIZE);
  //vector<float> labels = datasets::mnist::ReadLabels("../examples/mnist/t10k-labels-idx1-ubyte", numofdata, LABEL_SIZE);
  vector<float> images = datasets::mnist::ReadImages("../examples/mnist/train-images-idx3-ubyte", numofdata, IMAGE_SIZE);
  vector<float> labels = datasets::mnist::ReadLabels("../examples/mnist/train-labels-idx1-ubyte", numofdata, LABEL_SIZE);
  cerr << "images=" << images.size() << " labels=" << labels.size() << endl;
  cerr << "numofdata=" << numofdata << endl;

  size_t startInd = 0;
  size_t startIndData = 0;
  while (startInd < numofdata) {
	  size_t batchSize = (startInd + BATCH_SIZE < numofdata) ? BATCH_SIZE : numofdata - startInd;
	  cerr << "startInd=" << startInd
			  << " startIndData=" << startIndData
			  << " batchSize=" << batchSize << endl;

	  Tensor tx({numofdata, IMAGE_SIZE}, 1);
	  Tensor ty({numofdata, LABEL_SIZE}, 1);

	  tx.Load(images.begin() + startIndData, images.begin() + startIndData + batchSize * IMAGE_SIZE);
	  ty.Load(labels.begin() + startInd, labels.begin() + startInd + batchSize);

	  //cerr << "tx=" << Debug(tx.shape()) << endl;
	  //cerr << "ty=" << Debug(ty.shape()) << endl;

	  x = tx;
	  y = ty;

	  cerr << "x=" << Debug(x.val().shape()) << endl;
	  cerr << "y=" << Debug(y.val().shape()) << endl;


	  graph.forward(batchSize);

	  cerr << "w=" << Debug(w.val().shape()) << endl;
	  cerr << "b=" << Debug(b.val().shape()) << endl;
	  std::cerr << "z: " << Debug(z.val().shape()) << endl;
	  std::cerr << "lr: " << Debug(lr.val().shape()) << endl;
	  std::cerr << "Log-likelihood: " << Debug(graph.val().shape()) << endl ;

	  //std::cerr << "scores=" << scores.val().Debug() << endl;
	  //std::cerr << "lr=" << lr.val().Debug() << endl;

	  graph.backward();

	  //std::cerr << graph["pred"].val()[0] << std::endl;

	  startInd += batchSize;
	  startIndData += batchSize * IMAGE_SIZE;
  }
  

   // XOR
  /*
  Expr x = input(shape={whatevs, 2}, name="X");
  Expr y = input(shape={whatevs, 2}, name="Y");

  Expr w = param(shape={2, 1}, name="W0");
  Expr b = param(shape={1, 1}, name="b0");

  Expr n5 = dot(x, w);
  Expr n6 = n5 + b;
  Expr lr = softmax(n6, axis=1, name="pred");
  cerr << "lr=" << lr.Debug() << endl;

  Expr graph = -mean(sum(y * log(lr), axis=1), axis=0, name="cost");

  Tensor tx({4, 2}, 1);
  Tensor ty({4, 1}, 1);
  cerr << "tx=" << tx.Debug() << endl;
  cerr << "ty=" << ty.Debug() << endl;

  tx.Load("../examples/xor/train.txt");
  ty.Load("../examples/xor/label.txt");
  */

#if 0  
  hook0(graph);
  graph.autodiff();
  std::cerr << graph["cost"].val()[0] << std::endl;
  //hook1(graph);
  for(auto p : graph.params()) {
    auto update = _1 = _1 - alpha * _2;
    Element(update, p.val(), p.grad());
  }
  hook2(graph);
  
  auto opt = adadelta(cost_function=cost,
                      eta=0.9, gamma=0.1,
                      set_batch=set,
                      before_update=before,
                      after_update=after,
                      set_valid=valid,
                      validation_freq=100,
                      verbose=1, epochs=3, early_stopping=10);
  opt.run();
#endif  
  return 0;
}
