#include "hip/hip_runtime.h"

#include "marian.h"
#include "mnist.h"
#include "vocab.h"
#include <assert.h>

using namespace marian;
using namespace keywords;

const int input_size = 10;
const int output_size = 15;
const int batch_size = 25;
const int hidden_size = 5;
const int num_inputs = 8;
const int num_outputs = 6;

ExpressionGraph build_graph(int cuda_device) {
  std::cerr << "Building computation graph..." << std::endl;

  ExpressionGraph g(cuda_device);
  std::vector<Expr> X, Y, H, S;

  // For the stop symbol.
  for (int t = 0; t <= num_inputs; ++t) {
    std::stringstream ss;
    ss << "X" << t;
    X.emplace_back(named(g.input(shape={batch_size, input_size}), ss.str()));
  }

  // For the stop symbol.
  for (int t = 0; t <= num_outputs; ++t) {
    std::stringstream ss;
    ss << "Y" << t;
    Y.emplace_back(named(g.input(shape={batch_size, output_size}), ss.str()));
  }

  Expr Wxh = named(g.param(shape={input_size, hidden_size}, init=uniform()), "Wxh");
  Expr Whh = named(g.param(shape={hidden_size, hidden_size}, init=uniform()), "Whh");
  Expr bh = named(g.param(shape={1, hidden_size}, init=uniform()), "bh");
  Expr h0 = named(g.param(shape={1, hidden_size}, init=uniform()), "h0");

  std::cerr << "Building encoder RNN..." << std::endl;
  H.emplace_back(tanh(dot(X[0], Wxh) + dot(h0, Whh) + bh));
  for (int t = 1; t <= num_inputs; ++t) {
    H.emplace_back(tanh(dot(X[t], Wxh) + dot(H[t-1], Whh) + bh));
  }

  Expr Wxh_d = named(g.param(shape={output_size, hidden_size}, init=uniform()), "Wxh_d");
  Expr Whh_d = named(g.param(shape={hidden_size, hidden_size}, init=uniform()), "Whh_d");
  Expr bh_d = named(g.param(shape={1, hidden_size}, init=uniform()), "bh_d");

  std::cerr << "Building decoder RNN..." << std::endl;
  auto h0_d = H[num_inputs];
  S.emplace_back(tanh(dot(Y[0], Wxh_d) + dot(h0_d, Whh_d) + bh_d));
  for (int t = 1; t < num_outputs; ++t) {
    S.emplace_back(tanh(dot(Y[t], Wxh_d) + dot(S[t-1], Whh_d) + bh_d));
  }

  Expr Why = named(g.param(shape={hidden_size, output_size}, init=uniform()), "Why");
  Expr by = named(g.param(shape={1, output_size}, init=uniform()), "by");

  std::cerr << "Building output layer..." << std::endl;
  std::vector<Expr> Yp;

  Yp.emplace_back(named(softmax_fast(dot(h0_d, Why) + by), "pred"));
  Expr cross_entropy = sum(Y[0] * log(Yp[0]), axis=1);
  for (int t = 1; t <= num_outputs; ++t) {
    Yp.emplace_back(named(softmax_fast(dot(S[t-1], Why) + by), "pred"));
    cross_entropy = cross_entropy + sum(Y[t] * log(Yp[t]), axis=1);
  }
  auto cost = named(-mean(cross_entropy, axis=0), "cost");

  std::cerr << "Done." << std::endl;

  return g;
}



int main(int argc, char** argv) {
#if 1
  std::cerr << "Loading the data... ";
  Vocab sourceVocab, targetVocab;

  // read parallel corpus from file
  std::fstream sourceFile("../examples/mt/dev/newstest2013.de");
  std::fstream targetFile("../examples/mt/dev/newstest2013.en");

  std::vector<std::vector<size_t> > source_sentences, target_sentences;
  std::string sourceLine, targetLine;
  while (getline(sourceFile, sourceLine)) {
    getline(targetFile, targetLine);
    std::vector<size_t> sourceIds = sourceVocab.ProcessSentence(sourceLine);
    std::vector<size_t> targetIds = targetVocab.ProcessSentence(targetLine);
    source_sentences.push_back(sourceIds);
    target_sentences.push_back(targetIds);
  }
  std::cerr << "Done." << std::endl;
  std::cerr << source_sentences.size()
            << " sentence pairs read." << std::endl;
  std::cerr << "Source vocabulary size: " << sourceVocab.Size() << std::endl;
  std::cerr << "Target vocabulary size: " << targetVocab.Size() << std::endl;
#endif

  ExpressionGraph g = build_graph(0);

  // For the stop symbol.
  for (int t = 0; t <= num_inputs; ++t) {
    Tensor Xt({batch_size, input_size});

    float max = 1.;
    std::vector<float> values(batch_size * input_size);
    std::vector<float> classes(batch_size * output_size, 0.0);
    int k = 0;
    for (int i = 0; i < batch_size; ++i) {
      for (int j = 0; j < input_size; ++j, ++k) {
         values[k] = max * (2.0*static_cast<float>(rand()) / RAND_MAX - 1.0);
      }
    }

    thrust::copy(values.begin(), values.end(), Xt.begin());

    std::stringstream ss;
    ss << "X" << t;
    if (!g.has_node(ss.str())) std::cerr << "No node " << ss.str() << "!!!" << std::endl;
    g[ss.str()] = Xt;

  }

  for (int t = 0; t <= num_outputs; ++t) {
    Tensor Yt({batch_size, output_size});

    std::vector<float> classes(batch_size * output_size, 0.0);
    int l = 0;
    for (int i = 0; i < batch_size; ++i) {
      int gold = output_size * static_cast<float>(rand()) / RAND_MAX;
      classes[l + gold] = 1.0;
      l += output_size;
    }

    thrust::copy(classes.begin(), classes.end(), Yt.begin());

    std::stringstream ss;
    ss << "Y" << t;
    if (!g.has_node(ss.str())) std::cerr << "No node " << ss.str() << "!!!" << std::endl;
    g[ss.str()] = Yt;
  }

  std::cerr << "Graphviz step" << std::endl;
  std::cout << g.graphviz() << std::endl;

  std::cerr << "Forward step" << std::endl;
  g.forward(batch_size);
  std::cerr << "Backward step" << std::endl;
  g.backward();
  std::cerr << "Done" << std::endl;

  std::cerr << g["cost"].val().Debug() << std::endl;

  std::cerr << g["X0"].val().Debug() << std::endl;
  std::cerr << g["Y0"].val().Debug() << std::endl;

#if 1
  std::cerr << g["Whh"].grad().Debug() << std::endl;
  std::cerr << g["bh"].grad().Debug() << std::endl;
  std::cerr << g["Why"].grad().Debug() << std::endl;
  std::cerr << g["by"].grad().Debug() << std::endl;
  std::cerr << g["Wxh"].grad().Debug() << std::endl;
  std::cerr << g["h0"].grad().Debug() << std::endl;
#endif

  return 0;
}
