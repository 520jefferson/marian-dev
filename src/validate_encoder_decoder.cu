#include "hip/hip_runtime.h"

#include "marian.h"
#include "mnist.h"
#include "vocab.h"
#include <assert.h>

using namespace marian;
using namespace keywords;

ExpressionGraph build_graph(int source_vocabulary_size,
                            int target_vocabulary_size,
                            int embedding_size,
                            int hidden_size,
                            int num_source_tokens,
                            int num_target_tokens) {
  std::cerr << "Building computation graph..." << std::endl;

  int input_size = source_vocabulary_size;
  int output_size = target_vocabulary_size;
  int num_inputs = num_source_tokens;
  int num_outputs = num_target_tokens;

  ExpressionGraph g;
  std::vector<Expr> X, Y, H, S;

  // We're including the stop symbol here.
  for (int t = 0; t <= num_inputs; ++t) {
    std::stringstream ss;
    ss << "X" << t;
    X.emplace_back(named(g.input(shape={whatevs, input_size}), ss.str()));
  }

  // We're including the stop symbol here.
  for (int t = 0; t <= num_outputs; ++t) {
    std::stringstream ss;
    ss << "Y" << t;
    Y.emplace_back(named(g.input(shape={whatevs, output_size}), ss.str()));
  }

  // Source embeddings.
  Expr E = named(g.param(shape={input_size, embedding_size},
                         init=uniform()), "E");

  // Source RNN parameters.
  Expr Wxh = named(g.param(shape={embedding_size, hidden_size},
                   init=uniform(-0.1, 0.1)), "Wxh");
  Expr Whh = named(g.param(shape={hidden_size, hidden_size},
                   init=uniform(-0.1, 0.1)), "Whh");
  Expr bh = named(g.param(shape={1, hidden_size},
                  init=uniform(-0.1, 0.1)), "bh");
  Expr h0 = named(g.param(shape={1, hidden_size},
                  init=uniform(-0.1, 0.1)), "h0");

  std::cerr << "Building encoder RNN..." << std::endl;
  H.emplace_back(tanh(dot(dot(X[0], E), Wxh) + dot(h0, Whh) + bh));
  for (int t = 1; t <= num_inputs; ++t) {
    H.emplace_back(tanh(dot(dot(X[t], E), Wxh) + dot(H[t-1], Whh) + bh));
  }

  // Target RNN parameters.
  Expr Wxh_d = named(g.param(shape={output_size, hidden_size},
                     init=uniform(-0.1, 0.1)), "Wxh_d");
  Expr Whh_d = named(g.param(shape={hidden_size, hidden_size},
                     init=uniform(-0.1, 0.1)), "Whh_d");
  Expr bh_d = named(g.param(shape={1, hidden_size},
                    init=uniform(-0.1, 0.1)), "bh_d");

  std::cerr << "Building decoder RNN..." << std::endl;
  auto h0_d = H[num_inputs];
  S.emplace_back(tanh(dot(Y[0], Wxh_d) + dot(h0_d, Whh_d) + bh_d));
  for (int t = 1; t < num_outputs; ++t) {
    S.emplace_back(tanh(dot(Y[t], Wxh_d) + dot(S[t-1], Whh_d) + bh_d));
  }

  // Output linear layer before softmax.
  Expr Why = named(g.param(shape={hidden_size, output_size},
                           init=uniform(-0.1, 0.1)), "Why");
  Expr by = named(g.param(shape={1, output_size},
                          init=uniform(-0.1, 0.1)), "by");

  std::cerr << "Building output layer..." << std::endl;

  // Softmax layer and cost function.
  std::vector<Expr> Yp;
  Yp.emplace_back(named(softmax(dot(h0_d, Why) + by), "pred"));
  Expr cross_entropy = sum(Y[0] * log(Yp[0]), axis=1);
  for (int t = 1; t <= num_outputs; ++t) {
    Yp.emplace_back(named(softmax(dot(S[t-1], Why) + by), "pred"));
    cross_entropy = cross_entropy + sum(Y[t] * log(Yp[t]), axis=1);
  }
  auto cost = named(-mean(cross_entropy, axis=0), "cost");

  std::cerr << "Done." << std::endl;

  return g;
}

int main(int argc, char** argv) {
#if 1
  std::cerr << "Loading the data... ";
  Vocab source_vocab, target_vocab;

  // read parallel corpus from file
  std::fstream source_file("../examples/mt/dev/newstest2013.de");
  std::fstream target_file("../examples/mt/dev/newstest2013.en");

  // Right now we're only reading the first few sentence pairs, and defining
  // that as the step size.
  int batch_size = 64;
  int num_source_tokens = -1;
  int num_target_tokens = -1;
  std::vector<std::vector<size_t> > source_sentences, target_sentences;
  std::string source_line, target_line;
  while (getline(source_file, source_line)) {
    getline(target_file, target_line);
    std::vector<size_t> source_ids = source_vocab.ProcessSentence(source_line);
    source_ids.push_back(source_vocab.GetEOS()); // Append EOS token.
    std::vector<size_t> target_ids = target_vocab.ProcessSentence(target_line);
    target_ids.push_back(target_vocab.GetEOS()); // Append EOS token.
    source_sentences.push_back(source_ids);
    target_sentences.push_back(target_ids);
    if (num_source_tokens < 0 || source_ids.size() > num_source_tokens) {
      num_source_tokens = source_ids.size();
    }
    if (num_target_tokens < 0 || target_ids.size() > num_target_tokens) {
      num_target_tokens = target_ids.size();
    }
    if (source_sentences.size() == batch_size) break;
  }
  std::cerr << "Done." << std::endl;
  std::cerr << source_sentences.size()
            << " sentence pairs read." << std::endl;
  std::cerr << "Source vocabulary size: " << source_vocab.Size() << std::endl;
  std::cerr << "Target vocabulary size: " << target_vocab.Size() << std::endl;
  std::cerr << "Max source tokens: " << num_source_tokens << std::endl;
  std::cerr << "Max target tokens: " << num_target_tokens << std::endl;

  // Padding the source and target sentences.
  for (auto &sentence : source_sentences) {
    for (int i = sentence.size(); i < num_source_tokens; ++i) {
      sentence.push_back(source_vocab.GetPAD());
    }
  }
  for (auto &sentence : target_sentences) {
    for (int i = sentence.size(); i < num_target_tokens; ++i) {
      sentence.push_back(target_vocab.GetPAD());
    }
  }

  std::cerr << "Building the encoder-decoder computation graph..." << std::endl;

  // Build the encoder-decoder computation graph.
  int embedding_size = 50;
  int hidden_size = 100;
  ExpressionGraph g = build_graph(source_vocab.Size(),
                                  target_vocab.Size(),
                                  embedding_size,
                                  hidden_size,
                                  num_source_tokens-1,
                                  num_target_tokens-1);

  std::cerr << "Attaching the data to the computation graph..." << std::endl;

  // Convert the data to dense one-hot vectors.
  // TODO: make the graph handle sparse indices with a proper lookup layer.
  for (int t = 0; t < num_source_tokens; ++t) {
    Tensor Xt({batch_size, static_cast<int>(source_vocab.Size())});
    std::vector<float> values(batch_size * source_vocab.Size(), 0.0);
    int k = 0;
    for (int i = 0; i < batch_size; ++i) {
      values[k + source_sentences[i][t]] = 1.0;
      k += source_vocab.Size();
    }
    thrust::copy(values.begin(), values.end(), Xt.begin());
    // Attach this slice to the graph.
    std::stringstream ss;
    ss << "X" << t;
    g[ss.str()] = Xt;
  }

  for (int t = 0; t < num_target_tokens; ++t) {
    Tensor Yt({batch_size, static_cast<int>(target_vocab.Size())});
    std::vector<float> values(batch_size * target_vocab.Size(), 0.0);
    int k = 0;
    for (int i = 0; i < batch_size; ++i) {
      values[k + target_sentences[i][t]] = 1.0;
      k += target_vocab.Size();
    }
    thrust::copy(values.begin(), values.end(), Yt.begin());
    // Attach this slice to the graph.
    std::stringstream ss;
    ss << "Y" << t;
    g[ss.str()] = Yt;
  }

#else

  int source_vocabulary_size = 10;
  int target_vocabulary_size = 15;
  int embedding_size = 8;
  int hidden_size = 5;
  int batch_size = 25;
  int num_source_tokens = 8;
  int num_target_tokens = 6;

  // Build the encoder-decoder computation graph.
  ExpressionGraph g = build_graph(0, // cuda device.
                                  source_vocabulary_size,
                                  target_vocabulary_size,
                                  embedding_size,
                                  hidden_size,
                                  num_source_tokens,
                                  num_target_tokens);

  int input_size = source_vocabulary_size;
  int output_size = target_vocabulary_size;
  int num_inputs = num_source_tokens;
  int num_outputs = num_target_tokens;

  // Generate input data (include the stop symbol).
  for (int t = 0; t <= num_inputs; ++t) {
    Tensor Xt({batch_size, input_size});
    float max = 1.;
    std::vector<float> values(batch_size * input_size);
    std::vector<float> classes(batch_size * output_size, 0.0);
    int k = 0;
    for (int i = 0; i < batch_size; ++i) {
      for (int j = 0; j < input_size; ++j, ++k) {
         values[k] = max * (2.0*static_cast<float>(rand()) / RAND_MAX - 1.0);
      }
    }
    thrust::copy(values.begin(), values.end(), Xt.begin());
    std::stringstream ss;
    ss << "X" << t;
    g[ss.str()] = Xt;
  }

  // Generate output data (include the stop symbol).
  for (int t = 0; t <= num_outputs; ++t) {
    Tensor Yt({batch_size, output_size});

    std::vector<float> classes(batch_size * output_size, 0.0);
    int l = 0;
    for (int i = 0; i < batch_size; ++i) {
      int gold = output_size * static_cast<float>(rand()) / RAND_MAX;
      classes[l + gold] = 1.0;
      l += output_size;
    }
    thrust::copy(classes.begin(), classes.end(), Yt.begin());
    std::stringstream ss;
    ss << "Y" << t;
    g[ss.str()] = Yt;
  }
#endif

  std::cerr << "Printing the computation graph..." << std::endl;
  std::cout << g.graphviz() << std::endl;

  std::cerr << "Running the forward step..." << std::endl;
  g.forward(batch_size);
  std::cerr << "Running the backward step..." << std::endl;
  g.backward();
  std::cerr << "Done." << std::endl;

  std::cerr << g["cost"].val().Debug() << std::endl;

#if 0
  std::cerr << g["X0"].val().Debug() << std::endl;
  std::cerr << g["Y0"].val().Debug() << std::endl;
  std::cerr << g["Whh"].grad().Debug() << std::endl;
  std::cerr << g["bh"].grad().Debug() << std::endl;
  std::cerr << g["Why"].grad().Debug() << std::endl;
  std::cerr << g["by"].grad().Debug() << std::endl;
  std::cerr << g["Wxh"].grad().Debug() << std::endl;
  std::cerr << g["h0"].grad().Debug() << std::endl;
#endif

  return 0;
}
