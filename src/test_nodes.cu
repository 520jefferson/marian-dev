#include "hip/hip_runtime.h"
#include <vector>
#include <random>
#include "marian.h"
#include "expression_graph.h"
#include "keywords.h"
#include "definitions.h"


float Rand()
{
	float LO = -10;
	float HI = +20;
	float r3 = LO + static_cast <float> (rand()) /( static_cast <float> (RAND_MAX/(HI-LO)));
	return r3;
}

int main(int argc, char** argv)
{
  using namespace std;
  using namespace marian;
  using namespace keywords;

  int input_size = 10;
  int output_size = 10;
  int batch_size = 25;

  // define graph
  ExpressionGraph g;
  Expr inExpr = g.input(shape={batch_size, input_size});
  Expr labelExpr = g.input(shape={batch_size, output_size});

  //Expr outExpr = softmax(inExpr);
  Expr outExpr = tanh(inExpr);
  //Expr outExpr = - inExpr;
  Expr ceExpr = cross_entropy(outExpr, labelExpr);
  Expr cost = mean(ceExpr, axis=0);

  // create data
  srand(0);
  std::vector<float> values(batch_size * input_size);
  generate(begin(values), end(values), Rand);

  std::vector<float> labels(batch_size * input_size);
  generate(begin(labels), end(labels), Rand);

  Tensor inTensor({batch_size, input_size});
  thrust::copy(values.begin(), values.end(), inTensor.begin());

  Tensor labelTensor({batch_size, input_size});
  thrust::copy(labels.begin(), labels.end(), labelTensor.begin());

  inExpr = inTensor;
  labelExpr = labelTensor;

  // train
  g.forward(batch_size);
  //g.backward();
  g.backward_debug(0.00001);

  std::cout << g.graphviz() << std::endl;

  std::cerr << "inTensor=" << inTensor.Debug() << std::endl;

  Tensor outTensor = outExpr.val();
  std::cerr << "outTensor=" << outTensor.Debug() << std::endl;

  Tensor outGrad = outExpr.grad();
  std::cerr << "outGrad=" << outGrad.Debug() << std::endl;


}
