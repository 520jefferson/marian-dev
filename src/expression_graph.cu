#include <sstream>
#include "expression_graph.h"

using namespace std;

namespace marian {

Expr::Expr(ExpressionGraphPtr g, Chainable<Tensor>* chainable)
  : graph_(g), pimpl_(chainable) {
  graph_->stack()->push_back(chainable);    
}

Tensor Expr::val() {
  return pimpl_->val();
}

Tensor Expr::grad() {
    return pimpl_->grad();
}

ChainPtr Expr::node() {
    return pimpl_;
}

ExpressionGraphPtr Expr::graph() {
    return graph_;
}
  
Expr::operator ChainPtr() {
  return pimpl_;
}

std::string Expr::Debug() const
{
	stringstream strm;
	const Shape &shape = pimpl_->shape();
	strm << marian::Debug(shape);
	return strm.str();
}

///////////////////////////////////////////////////////
#include "tensor_operators.h"

void temp()
{
	std::vector<float> hVec({1,2,  4,3,  7,9,  7,3});
	thrust::device_vector<float> dVec(8);
	thrust::copy(hVec.begin(), hVec.end(), dVec.begin());
	float *data = thrust::raw_pointer_cast(dVec.data());

	gSoftMax<<<4, 2, sizeof(float)>>>(data, 4, 2);

	std::vector<float> hVec2(8);
	thrust::copy(dVec.begin(), dVec.end(), hVec2.begin());
	cerr << "hVec2=";
	for (size_t i = 0; i < hVec.size(); ++i) {
		cerr << hVec2[i] << " ";
	}
	cerr << endl;
}

ExpressionGraph::ExpressionGraph(int cudaDevice)
: stack_(new ChainableStack)
{
  std::srand (time(NULL));
  hipSetDevice(0);

  temp();
  exit(0);
}

}
