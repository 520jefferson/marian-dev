#include "hip/hip_runtime.h"

#include "marian.h"
#include "mnist.h"
#include "vocab.h"
#include <assert.h>

using namespace marian;
using namespace keywords;

const int input_size = 10;
const int output_size = 15;
const int embedding_size = 8;
const int hidden_size = 5;
const int batch_size = 25;
const int num_inputs = 8;
const int num_outputs = 6;

ExpressionGraph build_graph() {
  std::cerr << "Building computation graph..." << std::endl;

  ExpressionGraph g;
  std::vector<Expr> X, Y, H, S;

  // We're including the stop symbol here.
  for (int t = 0; t <= num_inputs; ++t) {
    std::stringstream ss;
    ss << "X" << t;
    X.emplace_back(named(g.input(shape={batch_size, input_size}), ss.str()));
  }

  // We're including the stop symbol here.
  for (int t = 0; t <= num_outputs; ++t) {
    std::stringstream ss;
    ss << "Y" << t;
    Y.emplace_back(named(g.input(shape={batch_size, output_size}), ss.str()));
  }

  // Source embeddings.
  Expr E = named(g.param(shape={input_size, embedding_size},
                         init=uniform()), "E");

  // Source RNN parameters.
  Expr Wxh = named(g.param(shape={embedding_size, hidden_size},
                   init=uniform()), "Wxh");
  Expr Whh = named(g.param(shape={hidden_size, hidden_size},
                   init=uniform()), "Whh");
  Expr bh = named(g.param(shape={1, hidden_size},
                  init=uniform()), "bh");
  Expr h0 = named(g.param(shape={1, hidden_size},
                  init=uniform()), "h0");

  std::cerr << "Building encoder RNN..." << std::endl;
  H.emplace_back(tanh(dot(dot(X[0], E), Wxh) + dot(h0, Whh) + bh));
  for (int t = 1; t <= num_inputs; ++t) {
    H.emplace_back(tanh(dot(dot(X[t], E), Wxh) + dot(H[t-1], Whh) + bh));
  }

  // Target RNN parameters.
  Expr Wxh_d = named(g.param(shape={output_size, hidden_size},
                     init=uniform()), "Wxh_d");
  Expr Whh_d = named(g.param(shape={hidden_size, hidden_size},
                     init=uniform()), "Whh_d");
  Expr bh_d = named(g.param(shape={1, hidden_size},
                    init=uniform()), "bh_d");

  std::cerr << "Building decoder RNN..." << std::endl;
  auto h0_d = H[num_inputs];
  S.emplace_back(tanh(dot(Y[0], Wxh_d) + dot(h0_d, Whh_d) + bh_d));
  for (int t = 1; t < num_outputs; ++t) {
    S.emplace_back(tanh(dot(Y[t], Wxh_d) + dot(S[t-1], Whh_d) + bh_d));
  }

  // Output linear layer before softmax.
  Expr Why = named(g.param(shape={hidden_size, output_size},
                           init=uniform()), "Why");
  Expr by = named(g.param(shape={1, output_size},
                          init=uniform()), "by");

  std::cerr << "Building output layer..." << std::endl;

  // Softmax layer and cost function.
  std::vector<Expr> Yp;
  Yp.emplace_back(named(softmax(dot(h0_d, Why) + by), "pred"));
  Expr cross_entropy = sum(Y[0] * log(Yp[0]), axis=1);
  for (int t = 1; t <= num_outputs; ++t) {
    Yp.emplace_back(named(softmax(dot(S[t-1], Why) + by), "pred"));
    cross_entropy = cross_entropy + sum(Y[t] * log(Yp[t]), axis=1);
  }
  auto cost = named(-mean(cross_entropy, axis=0), "cost");

  std::cerr << "Done." << std::endl;

  return g;
}

int main(int argc, char** argv) {
#if 1
  std::cerr << "Loading the data... ";
  Vocab sourceVocab, targetVocab;

  // read parallel corpus from file
  std::fstream sourceFile("../examples/mt/dev/newstest2013.de");
  std::fstream targetFile("../examples/mt/dev/newstest2013.en");

  std::vector<std::vector<size_t> > source_sentences, target_sentences;
  std::string sourceLine, targetLine;
  while (getline(sourceFile, sourceLine)) {
    getline(targetFile, targetLine);
    std::vector<size_t> sourceIds = sourceVocab.ProcessSentence(sourceLine);
    std::vector<size_t> targetIds = targetVocab.ProcessSentence(targetLine);
    source_sentences.push_back(sourceIds);
    target_sentences.push_back(targetIds);
  }
  std::cerr << "Done." << std::endl;
  std::cerr << source_sentences.size()
            << " sentence pairs read." << std::endl;
  std::cerr << "Source vocabulary size: " << sourceVocab.Size() << std::endl;
  std::cerr << "Target vocabulary size: " << targetVocab.Size() << std::endl;
#endif

  // Build the encoder-decoder computation graph.
  ExpressionGraph g = build_graph();

  // Generate input data (include the stop symbol).
  for (int t = 0; t <= num_inputs; ++t) {
    Tensor Xt({batch_size, input_size});
    float max = 1.;
    std::vector<float> values(batch_size * input_size);
    std::vector<float> classes(batch_size * output_size, 0.0);
    int k = 0;
    for (int i = 0; i < batch_size; ++i) {
      for (int j = 0; j < input_size; ++j, ++k) {
         values[k] = max * (2.0*static_cast<float>(rand()) / RAND_MAX - 1.0);
      }
    }
    thrust::copy(values.begin(), values.end(), Xt.begin());
    std::stringstream ss;
    ss << "X" << t;
    g[ss.str()] = Xt;
  }

  // Generate output data (include the stop symbol).
  for (int t = 0; t <= num_outputs; ++t) {
    Tensor Yt({batch_size, output_size});

    std::vector<float> classes(batch_size * output_size, 0.0);
    int l = 0;
    for (int i = 0; i < batch_size; ++i) {
      int gold = output_size * static_cast<float>(rand()) / RAND_MAX;
      classes[l + gold] = 1.0;
      l += output_size;
    }
    thrust::copy(classes.begin(), classes.end(), Yt.begin());
    std::stringstream ss;
    ss << "Y" << t;
    g[ss.str()] = Yt;
  }

  std::cerr << "Printing the computation graph..." << std::endl;
  std::cout << g.graphviz() << std::endl;

  std::cerr << "Running the forward step..." << std::endl;
  g.forward(batch_size);
  std::cerr << "Running the backward step..." << std::endl;
  g.backward();
  std::cerr << "Done." << std::endl;

  std::cerr << g["cost"].val().Debug() << std::endl;

  std::cerr << g["X0"].val().Debug() << std::endl;
  std::cerr << g["Y0"].val().Debug() << std::endl;
  std::cerr << g["Whh"].grad().Debug() << std::endl;
  std::cerr << g["bh"].grad().Debug() << std::endl;
  std::cerr << g["Why"].grad().Debug() << std::endl;
  std::cerr << g["by"].grad().Debug() << std::endl;
  std::cerr << g["Wxh"].grad().Debug() << std::endl;
  std::cerr << g["h0"].grad().Debug() << std::endl;

  return 0;
}
