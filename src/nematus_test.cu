#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <iomanip>
#include <string>
#include <cstdio>
#include <boost/timer/timer.hpp>

#include "marian.h"
#include "nematus.h"
#include "batch_generator.h"
#include "optimizers.h"

#include "corpus.h"

int main(int argc, char** argv) {
  using namespace marian;
  using namespace data;

  hipSetDevice(0);

  Corpus corpus({"../test/newstest2015-deen-src.de",
                 "../test/newstest2015-deen-src.de",
                 "../test/newstest2015-deen-ref.en"}, {});

  auto nematus = New<Nematus>();
  nematus->reserveWorkspaceMB(2048);
  //nematus->load("../test/model.npz");

  auto opt = Optimizer<Adam>(0.0001
                             /*,clip=norm(1)*/);

  size_t batchSize = 3;
  float sum = 0;
  boost::timer::cpu_timer timer;
  for(int i = 1; i <= 2000; ++i) {

    // fake batch
    auto srcBatch = generateSrcBatch(batchSize);
    auto trgBatch = generateTrgBatch(batchSize);
    nematus->construct(srcBatch, trgBatch);

    opt->update(nematus);

    float cost = nematus->cost();
    sum += cost;

    //if(i % 1 == 0)
    //  std::cerr << ".";
    if(i % 1 == 0)
      std::cout << "[" << i << "]" << std::fixed << std::setfill(' ') << std::setw(9)
                << " - cost: " << cost << "/" << sum / i
                << " - time: " << timer.format(5, "%ws") << std::endl;
  }
  std::cout << std::endl;
  std::cout << timer.format(5, "%ws") << std::endl;

  nematus->save("../test/model.marian.npz");

  return 0;
}
