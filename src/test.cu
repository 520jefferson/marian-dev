#include "hip/hip_runtime.h"

#include "marian.h"
#include "mnist.h"

using namespace std;

int main(int argc, char** argv) {
  /*auto images = datasets::mnist::ReadImages("../examples/mnist/t10k-images-idx3-ubyte");*/
  /*auto labels = datasets::mnist::ReadLabels("../examples/mnist/t10k-labels-idx1-ubyte");*/
  /*std::cerr << images.size() << " " << images[0].size() << std::endl;*/

  using namespace marian;
  using namespace keywords;
  

  Expr x = input(shape={whatevs, 784}, name="X");
  Expr y = input(shape={whatevs, 10}, name="Y");
  
  Expr w = param(shape={784, 10}, name="W0");
  Expr b = param(shape={1, 10}, name="b0");
  
  auto scores = dot(x, w) + b;
  auto lr = softmax(scores, axis=1, name="pred");
  auto graph = -mean(sum(y * log(lr), axis=1), axis=0, name="cost");
  cerr << "lr=" << lr.Debug() << endl;

  
  Tensor tx({500, 784}, 1);
  Tensor ty({500, 10}, 1);

  int numImg, imgSize;
  vector<float> images = datasets::mnist::ReadImages("../examples/mnist/t10k-images-idx3-ubyte", numImg, imgSize);
  vector<float> labels = datasets::mnist::ReadLabels("../examples/mnist/t10k-labels-idx1-ubyte");
  cerr << "images=" << images.size() << " labels=" << labels.size() << endl;
  tx.Load(images);
  //ty.Load(labels);

  cerr << "tx=" << tx.Debug() << endl;
  cerr << "ty=" << ty.Debug() << endl;

  x = tx;
  y = ty;

  graph.forward(500);

  std::cerr << "Result: ";
  for (auto val : scores.val().shape()) {
    std::cerr << val << " ";
  }
  std::cerr << std::endl;
  std::cerr << "Result: ";
  for (auto val : lr.val().shape()) {
    std::cerr << val << " ";
  }
  std::cerr << std::endl;
  std::cerr << "Log-likelihood: ";
  for (auto val : graph.val().shape()) {
    std::cerr << val << " ";
  }
  std::cerr << std::endl;

  graph.backward();
  
  //std::cerr << graph["pred"].val()[0] << std::endl;
  

   // XOR
  /*
  Expr x = input(shape={whatevs, 2}, name="X");
  Expr y = input(shape={whatevs, 2}, name="Y");

  Expr w = param(shape={2, 1}, name="W0");
  Expr b = param(shape={1, 1}, name="b0");

  Expr n5 = dot(x, w);
  Expr n6 = n5 + b;
  Expr lr = softmax(n6, axis=1, name="pred");
  cerr << "lr=" << lr.Debug() << endl;

  Expr graph = -mean(sum(y * log(lr), axis=1), axis=0, name="cost");

  Tensor tx({4, 2}, 1);
  Tensor ty({4, 1}, 1);
  cerr << "tx=" << tx.Debug() << endl;
  cerr << "ty=" << ty.Debug() << endl;

  tx.Load("../examples/xor/train.txt");
  ty.Load("../examples/xor/label.txt");
  */

#if 0  
  hook0(graph);
  graph.autodiff();
  std::cerr << graph["cost"].val()[0] << std::endl;
  //hook1(graph);
  for(auto p : graph.params()) {
    auto update = _1 = _1 - alpha * _2;
    Element(update, p.val(), p.grad());
  }
  hook2(graph);
  
  auto opt = adadelta(cost_function=cost,
                      eta=0.9, gamma=0.1,
                      set_batch=set,
                      before_update=before,
                      after_update=after,
                      set_valid=valid,
                      validation_freq=100,
                      verbose=1, epochs=3, early_stopping=10);
  opt.run();
#endif  
  return 0;
}
