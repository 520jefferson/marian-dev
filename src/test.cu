#include "hip/hip_runtime.h"

#include "marian.h"
#include "mnist.h"

int main(int argc, char** argv) {
  hipSetDevice(0);

  using namespace marian;
  using namespace keywords;

  int input_size = 10;
  int output_size = 2;
  int batch_size = 25;
  int hidden_size = 5;
  int num_inputs = 8;

  std::vector<Expr*> X(num_inputs);
  std::vector<Expr*> Y(num_inputs);
  std::vector<Expr*> H(num_inputs);

  for (int t = 0; t < num_inputs; ++t) {
    X[t] = new Expr(input(shape={batch_size, input_size}));
    Y[t] = new Expr(input(shape={batch_size, output_size}));
  }

  Expr Wxh = param(shape={input_size, hidden_size}, init=uniform(), name="Wxh");
  Expr Whh = param(shape={hidden_size, hidden_size}, init=uniform(), name="Whh");
  Expr bh = param(shape={1, hidden_size}, init=uniform(), name="bh");
  Expr h0 = param(shape={1, hidden_size}, init=uniform(), name="h0");

  std::cerr << "Building RNN..." << std::endl;
  H[0] = new Expr(tanh(dot(*X[0], Wxh) + dot(h0, Whh) + bh));
  for (int t = 1; t < num_inputs; ++t) {
    H[t] = new Expr(tanh(dot(*X[t], Wxh) + dot(*H[t-1], Whh) + bh));
  }

  Expr Why = param(shape={hidden_size, output_size}, init=uniform(), name="Why");
  Expr by = param(shape={1, output_size}, init=uniform(), name="by");

  std::cerr << "Building output layer..." << std::endl;
  std::vector<Expr*> Yp(num_inputs);

  Expr* cross_entropy = NULL;
  for (int t = 0; t < num_inputs; ++t) {
    Yp[t] = new Expr(softmax_fast(dot(*H[t], Why) + by, name="pred"));
    if (!cross_entropy) {
      cross_entropy = new Expr(sum(*Y[t] * log(*Yp[t]), axis=1));
    } else {
      *cross_entropy = *cross_entropy + sum(*Y[t] * log(*Yp[t]), axis=1);
    }
  }
  auto graph = -mean(*cross_entropy, axis=0, name="cost");

  for (int t = 0; t < num_inputs; ++t) {
    Tensor Xt({batch_size, input_size});
    Tensor Yt({batch_size, output_size});

    float max = 1.;
    std::vector<float> values(batch_size * input_size);
    std::vector<float> classes(batch_size * output_size, 0.0);
    int k = 0;
    int l = 0;
    for (int i = 0; i < batch_size; ++i) {
      for (int j = 0; j < input_size; ++j, ++k) {
         values[k] = max * (2.0*static_cast<float>(rand()) / RAND_MAX - 1.0);
      }
      int gold = output_size * static_cast<float>(rand()) / RAND_MAX;
      classes[l + gold] = 1.0;
      l += output_size;
    }

    thrust::copy(values.begin(), values.end(), Xt.begin());
    thrust::copy(classes.begin(), classes.end(), Yt.begin());

    *X[t] = Xt;
    *Y[t] = Yt;
  }

  graph.forward(batch_size);
  graph.backward();

  std::cerr << graph.val().Debug() << std::endl;

  std::cerr << X[0]->val().Debug() << std::endl;
  std::cerr << Y[0]->val().Debug() << std::endl;

  std::cerr << Whh.grad().Debug() << std::endl;
  std::cerr << bh.grad().Debug() << std::endl;
  std::cerr << Why.grad().Debug() << std::endl;
  std::cerr << by.grad().Debug() << std::endl;
  std::cerr << Wxh.grad().Debug() << std::endl;
  std::cerr << h0.grad().Debug() << std::endl;

  return 0;
}
