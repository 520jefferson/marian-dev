#include <algorithm>
#include <chrono>
#include <iomanip>
#include <string>
#include <cstdio>
#include <boost/timer/timer.hpp>

#include "marian.h"
#include "mnist.h"
#include "trainer.h"
#include "models/feedforward.h"

#include "tensors/tensor.h"
#include "tensors/tensor_gpu.h"
#include "tensors/tensor_allocator.h"

using namespace marian;
using namespace keywords;
using namespace data;
using namespace models;

int main(int argc, char** argv) {
  hipdnnHandle_t cudnnHandle;
  hipdnnCreate(&cudnnHandle);

  hipdnnRNNDescriptor_t rnnDesc;
  hipdnnCreateRNNDescriptor(&rnnDesc);

  int rows = 1;
  int cols = 500;

  int hiddenSize = 1024;
  int seqLength = 10;

  TensorAllocator alloc = newTensorAllocator<DeviceGPU>();
  std::vector<Tensor> tensors(seqLength);
  for(int i = 0; i < seqLength; ++i) {
    alloc->allocate(tensors[i], {rows, cols});
    tensors[i]->set(i);
  }

  hipdnnTensorDescriptor_t xDesc[seqLength];
  for(int i = 0; i < seqLength; ++i) {
    xDesc[i] = std::static_pointer_cast<TensorGPU>(tensors[i])->cudnn();
  }

  // dropout
  float dropout = 0.0;
  hipdnnDropoutDescriptor_t dropDesc;
  size_t statesSize;
  void* states;
  hipdnnDropoutGetStatesSize(cudnnHandle, &statesSize);
  hipMalloc((void**)&states, statesSize);
  hipdnnCreateDropoutDescriptor(&dropDesc);
  hipdnnSetDropoutDescriptor(
    dropDesc,
    cudnnHandle,
    dropout,
    (void*)states,
    statesSize,
    1234);
  // dropout

  hipdnnSetRNNDescriptor(
    rnnDesc,
    hiddenSize,
    1,
    dropDesc,
    HIPDNN_LINEAR_INPUT,
    HIPDNN_UNIDIRECTIONAL,
    HIPDNN_RNN_TANH,
    HIPDNN_DATA_FLOAT);

  size_t workSpaceSize;
  hipdnnGetRNNWorkspaceSize(
    cudnnHandle,
    rnnDesc,
    seqLength,
    xDesc,
    &workSpaceSize);

  std::cerr << workSpaceSize << std::endl;

  size_t trainingReserveSize;
  hipdnnGetRNNTrainingReserveSize(
    cudnnHandle,
    rnnDesc,
    seqLength,
    xDesc,
    &trainingReserveSize);

  std::cerr << trainingReserveSize << std::endl;

  hipdnnDestroyRNNDescriptor(rnnDesc);

  return 0;
}
