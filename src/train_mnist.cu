#include "hip/hip_runtime.h"

#include "marian.h"
#include "mnist.h"
#include "optimizers.h"

int main(int argc, char** argv) {
  const size_t IMAGE_SIZE = 784;
  const size_t LABEL_SIZE = 10;
  int numofdata;

  std::vector<float> trainImages = datasets::mnist::ReadImages("../examples/mnist/t10k-images-idx3-ubyte", numofdata, IMAGE_SIZE);
  std::vector<float> trainLabels = datasets::mnist::ReadLabels("../examples/mnist/t10k-labels-idx1-ubyte", numofdata, LABEL_SIZE);

  using namespace marian;
  using namespace keywords;

  ExpressionGraph g;
  
  Expr x = named(g.input(shape={whatevs, IMAGE_SIZE}), "x");
  Expr y = named(g.input(shape={whatevs, LABEL_SIZE}), "y");

  Expr w = named(g.param(shape={IMAGE_SIZE, LABEL_SIZE}), "w");
  Expr b = named(g.param(shape={1, LABEL_SIZE}), "b");

  auto scores = dot(x, w) + b;
  auto lr = softmax(scores);
  auto cost = named(-mean(sum(y * log(lr), axis=1), axis=0), "cost");
  std::cerr << "lr=" << lr.Debug() << std::endl;

  Adagrad opt;
  opt(g, 300);
  
  return 0;
}
