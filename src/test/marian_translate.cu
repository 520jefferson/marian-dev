#include <algorithm>
#include <chrono>
#include <iomanip>
#include <string>
#include <cstdio>
#include <boost/timer/timer.hpp>
#include <boost/chrono.hpp>

#include "marian.h"
#include "training/config.h"
#include "optimizers/optimizers.h"
#include "optimizers/clippers.h"
#include "data/batch_generator.h"
#include "data/corpus.h"
#include "models/dl4mt.h"
#include "translator/nth_element.h"

namespace marian {

template <class Builder>
class BeamSearch {
  private:
    Ptr<Builder> builder_;
    size_t beamSize_;
    hipStream_t stream_{0};

  public:
    BeamSearch(Ptr<Builder> builder)
     : builder_(builder),
       beamSize_(1)
    {}

    void search(Ptr<ExpressionGraph> graph,
                Ptr<data::CorpusBatch> batch) {

      auto nth = New<NthElement>(beamSize_, batch->size(), stream_);

      Expr startState, hyps, probs;
      startState = builder_->buildEncoder(graph, batch);
      hyps = startState;

      std::vector<unsigned> outKeys;

      size_t pos = 0;
      bool first = true;

      size_t dimTrgVoc = 1;

      while(first || outKeys[0] != 0) {

        std::vector<size_t> hypIdx;
        std::vector<size_t> embIdx;
        for(auto k : outKeys) {
          hypIdx.push_back(k / dimTrgVoc);
          embIdx.push_back(k % dimTrgVoc);
        }

        std::tie(hyps, probs) = builder_->stepPredict(hyps, hypIdx, embIdx);
        pos = graph->forward(pos);

        std::cerr << hyps->val()->debug() << std::endl;
        std::cerr << probs->val()->debug() << std::endl;

        dimTrgVoc = probs->shape()[0];

        std::vector<float> outCosts;
        std::vector<size_t> beamSizes(batch->size(), beamSize_);

        outKeys.clear();
        nth->getNBestList(beamSizes, probs->val(),
                          outCosts, outKeys, first);
        first = false;

        for(int i = 0; i < outKeys.size(); ++i)
          std::cerr << i << " " << outKeys[i]
            << " " << outCosts[i] << std::endl;
      }

    }
};

}

int main(int argc, char** argv) {
  using namespace marian;
  using namespace data;

  auto options = New<Config>(argc, argv, false);

  std::vector<std::string> files =
    {"../test/mini.en"};

  std::vector<std::string> vocab =
    {"../benchmark/marian32K/train.tok.true.bpe.en.json"};

  YAML::Node& c = options->get();
  c["train-sets"] = files;
  c["vocabs"] = vocab;

  auto corpus = DataSet<Corpus>(options);
  BatchGenerator<Corpus> bg(corpus, options);

  auto graph = New<ExpressionGraph>();
  graph->setDevice(0);

  auto dl4mt = New<DL4MT>(options);
  dl4mt->load(graph, "../benchmark/marian32K/modelBN.90000.npz");

  graph->reserveWorkspaceMB(128);

  boost::timer::cpu_timer timer;
  bg.prepare(false);
  while(bg) {
    auto batch = bg.next();
    batch->debug();

    auto search = New<BeamSearch<DL4MT>>(dl4mt);
    search->search(graph, batch);

    exit(0);
  }
  std::cout << std::endl;
  std::cout << timer.format(5, "%ws") << std::endl;

  return 0;
}
