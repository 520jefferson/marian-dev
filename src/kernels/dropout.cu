#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "kernels/dropout.h"


#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    exit(1);}} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    exit(1);}} while(0)

namespace marian {

hiprandGenerator_t createCurandGenerator(size_t device,
                                        size_t seed) {
  hipSetDevice(device);
  hiprandGenerator_t generator;
  CURAND_CALL(hiprandCreateGenerator(&generator,
                                    HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(generator, seed));

  //hipStream_t stream = 0;
  //CURAND_CALL(hiprandSetStream(generator, stream));
  //CURAND_CALL(hiprandDestroyGenerator(generator));
  return generator;
}


__global__
void gScale(float* data, int n, float p) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  while (index < n) {
    data[index] = (data[index] < p) / p;
    index += gridDim.x * blockDim.x;
  }
}

void Dropout(Tensor tensor, float p,
             hiprandGenerator_t gen) {

  int n = tensor->size();
  CURAND_CALL(hiprandGenerateUniform(gen, tensor->data(), n));

  int numThreads = std::min(n, 512);
  int numBlocks = n / numThreads + (n % numThreads != 0);

  gScale<<<numBlocks, numThreads>>>(tensor->data(), n, 1.f - p);
}

}
