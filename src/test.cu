#include "hip/hip_runtime.h"

#include "marian.h"

using namespace std;

int main(int argc, char** argv) {

  using namespace marian;
  using namespace keywords;
  
  Expr x = input(shape={whatevs, 784}, name="X");
  Expr y = input(shape={whatevs, 10}, name="Y");
  
  Expr w = param(shape={784, 10}, name="W0");
  Expr b = param(shape={1, 10}, name="b0");
  
  Expr lr = softmax(dot(x, w) + b, axis=1, name="pred");
  Expr graph = -mean(sum(y * log(lr), axis=1), axis=0, name="cost");
  
  Tensor tx({500, 784}, 1);
  Tensor ty({500, 10}, 1);
  cerr << "tx=" << tx.Debug();
  cerr << "ty=" << ty.Debug();

  x = tx;
  y = ty;

  graph.forward(500);
  //std::cerr << graph["pred"].val()[0] << std::endl;
  
  
  //hook0(graph);
  //graph.autodiff();
  //std::cerr << graph["cost"].val()[0] << std::endl;
  //hook1(graph);
  //for(auto p : graph.params()) {
  //  auto update = _1 = _1 - alpha * _2;
  //  Element(update, p.val(), p.grad());
  //}
  //hook2(graph);
  //
  //auto opt = adadelta(cost_function=cost,
  //                    eta=0.9, gamma=0.1,
  //                    set_batch=set,
  //                    before_update=before,
  //                    after_update=after,
  //                    set_valid=valid,
  //                    validation_freq=100,
  //                    verbose=1, epochs=3, early_stopping=10);
  //opt.run();
  
  return 0;
}
