#include "hip/hip_runtime.h"

#include "marian.h"
#include "mnist.h"

using namespace std;

int main(int argc, char** argv) {
  /*int numImg = 0;*/
  /*auto images = datasets::mnist::ReadImages("../examples/mnist/t10k-images-idx3-ubyte", numImg);*/
  /*auto labels = datasets::mnist::ReadLabels("../examples/mnist/t10k-labels-idx1-ubyte", numImg);*/

  using namespace marian;
  using namespace keywords;
  
  const size_t IMAGE_SIZE = 784;
  const size_t LABEL_SIZE = 10;

  Expr x = input(shape={whatevs, IMAGE_SIZE}, name="X");
  Expr y = input(shape={whatevs, LABEL_SIZE}, name="Y");
  
  Expr w = param(shape={IMAGE_SIZE, LABEL_SIZE}, name="W0");
  Expr b = param(shape={1, LABEL_SIZE}, name="b0");
  
  auto z = dot(x, w) + b;
  auto pred = softmax(z);
  //auto decision = argmax(pred, axis=1);
  
  auto cost = -mean(sum(y * log(pred), axis=1),
                    axis=0);
  
  cerr << "pred=" << pred.Debug() << endl;

#if 0
  int numofdata;
  vector<float> images = datasets::mnist::ReadImages("../examples/mnist/t10k-images-idx3-ubyte", numofdata, IMAGE_SIZE);
  vector<float> labels = datasets::mnist::ReadLabels("../examples/mnist/t10k-labels-idx1-ubyte", numofdata, LABEL_SIZE);
  cerr << "images=" << images.size() << " labels=" << labels.size() << endl;
  cerr << "numofdata=" << numofdata << endl;

  Tensor tx({numofdata, IMAGE_SIZE}, 1);
  Tensor ty({numofdata, LABEL_SIZE}, 1);

  tx.Load(images);
  ty.Load(labels);

  cerr << "tx=" << tx.Debug() << endl;
  cerr << "ty=" << ty.Debug() << endl;
#else
  Tensor tx({500, 784}, 1);
  Tensor ty({500, 10}, 1);
#endif

  x = tx;
  y = ty;

  cost.forward(500);

  std::cerr << "Result: ";
  for (auto val : pred.val().shape()) {
    std::cerr << val << " ";
  }
  std::cerr << std::endl;
  std::cerr << "Result: ";
  for (auto val : pred.val().shape()) {
    std::cerr << val << " ";
  }
  std::cerr << std::endl;
  pred.val().Print();
  std::cerr << "Log-likelihood: ";
  for (auto val : cost.val().shape()) {
    std::cerr << val << " ";
  }
  std::cerr << std::endl;
  cost.val().Print();
  
  cost.backward();
  
  //std::cerr << graph["pred"].val()[0] << std::endl;
  

   // XOR
  /*
  Expr x = input(shape={whatevs, 2}, name="X");
  Expr y = input(shape={whatevs, 2}, name="Y");

  Expr w = param(shape={2, 1}, name="W0");
  Expr b = param(shape={1, 1}, name="b0");

  Expr n5 = dot(x, w);
  Expr n6 = n5 + b;
  Expr lr = softmax(n6, axis=1, name="pred");
  cerr << "lr=" << lr.Debug() << endl;

  Expr graph = -mean(sum(y * log(lr), axis=1), axis=0, name="cost");

  Tensor tx({4, 2}, 1);
  Tensor ty({4, 1}, 1);
  cerr << "tx=" << tx.Debug() << endl;
  cerr << "ty=" << ty.Debug() << endl;

  tx.Load("../examples/xor/train.txt");
  ty.Load("../examples/xor/label.txt");
  */

#if 0  
  hook0(graph);
  graph.autodiff();
  std::cerr << graph["cost"].val()[0] << std::endl;
  //hook1(graph);
  for(auto p : graph.params()) {
    auto update = _1 = _1 - alpha * _2;
    Element(update, p.val(), p.grad());
  }
  hook2(graph);
  
  auto opt = adadelta(cost_function=cost,
                      eta=0.9, gamma=0.1,
                      set_batch=set,
                      before_update=before,
                      after_update=after,
                      set_valid=valid,
                      validation_freq=100,
                      verbose=1, epochs=3, early_stopping=10);
  opt.run();
#endif  
  return 0;
}
