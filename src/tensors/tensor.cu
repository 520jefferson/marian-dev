#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

#include "kernels/cuda_helpers.h"
#include "kernels/tensor_operators.h"
#include "tensors/tensor.h"

namespace marian {

template <typename T>
__global__ void gFill(T *d_in, int size, T val) {
  for(int bid = 0; bid < size; bid += blockDim.x * gridDim.x) {
    int index = bid + threadIdx.x + blockDim.x * blockIdx.x;
    if(index < size) {
      d_in[index] = val;
    }
  }
}

void TensorBase::get(void* value, size_t sizeOf, size_t num, size_t offset) {
  hipSetDevice(device_);
  CUDA_CHECK(
      hipMemcpy(value,
                 memory_->data() + offset * sizeOf,
                 num * sizeOf,
                 hipMemcpyDeviceToHost));
  hipStreamSynchronize(0);
}

void TensorBase::set(const void* value, size_t sizeOf, size_t num, size_t offset) {
  hipSetDevice(device_);
  CUDA_CHECK(
      hipMemcpy(memory_->data() + offset * sizeOf,
                 value,
                 num * sizeOf,
                 hipMemcpyHostToDevice));
  hipStreamSynchronize(0);
}

void TensorBase::set(float value) {
  hipSetDevice(device_);
  int threads = std::min(512, (int)size());
  int blocks = (size() / threads) + (size() % threads != 0);
  gFill<<<blocks, threads>>>(data<float>(), size(), value);
  hipStreamSynchronize(0);
}

void TensorBase::setSparse(const std::vector<size_t> &k,
                           const std::vector<float> &v) {
  hipSetDevice(device_);
  SetSparse(data(), k, v);
  hipStreamSynchronize(0);
}

void TensorBase::copyFrom(Tensor in) {
  hipSetDevice(device_);
  CUDA_CHECK(hipMemcpy(
      memory()->data(), in->memory()->data(), in->memory()->size(), hipMemcpyDefault));
  hipStreamSynchronize(0);
}

Tensor operator<<(Tensor t, const std::vector<float> &v) {
  t->set(v);
  return t;
}

Tensor operator>>(Tensor t, std::vector<float> &v) {
  t->get(v);
  return t;
}
}
