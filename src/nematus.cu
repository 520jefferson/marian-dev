#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <iomanip>
#include <string>
#include <cstdio>
#include <boost/timer/timer.hpp>
#include <boost/chrono.hpp>

#include "marian.h"
#include "nematus.h"
#include "batch_generator.h"
#include "optimizers.h"

#include "corpus.h"

int main(int argc, char** argv) {
  using namespace marian;
  using namespace data;

  hipSetDevice(0);

  std::vector<std::string> files =
    {"../train.src-pe.gpu0/train.all.src",
     "../train.src-pe.gpu0/train.all.pe"};

  std::vector<std::string> vocab =
    {"../train.src-pe.gpu0/src.json",
     "../train.src-pe.gpu0/pe.json"};

  /*
  std::vector<std::string> files =
    {"/work/wmt16/work/unbabel/wmt2015/APE/train.mt-pe.gpu0/train.all.mt",
     "/work/wmt16/work/unbabel/wmt2015/APE/train.mt-pe.gpu0/train.all.pe"};

  std::vector<std::string> vocab =
    {"/work/wmt16/work/unbabel/wmt2015/APE/train.mt-pe.gpu0/mt.json",
     "/work/wmt16/work/unbabel/wmt2015/APE/train.mt-pe.gpu0/pe.json"};
  */

  auto corpus = DataSet<Corpus>(files, vocab, 50);
  BatchGenerator<Corpus> bg(corpus, 40, 1000);

  auto nematus = New<Nematus>();
  nematus->load("../train.src-pe.gpu0/model.iter10000.npz");
  nematus->reserveWorkspaceMB(8000);

  auto opt = Optimizer<Adam>(0.0001 /*, clip=norm(1)*/);

  float sum = 0;
  boost::timer::cpu_timer timer;
  size_t batches = 1;
  for(int i = 0; i < 20; ++i) {
    bg.prepare();
    while(bg) {
      auto batch = bg.next();

      nematus->construct(*batch);

      opt->update(nematus);

      float cost = nematus->cost();
      sum += cost;

      if(batches % 100 == 0) {
        std::cout << std::setfill(' ')
                  << "Epoch " << i
                  << " Update " << batches
                  << " Cost "   << std::setw(7) << std::setprecision(6) << cost
                  << " UD " << timer.format(2, "%ws");

        float seconds = std::stof(timer.format(5, "%w"));
        float sentences = 100 * batch->size() / seconds;

        std::cout << " " << std::setw(5)
                  << std::setprecision(4)
                  << sentences
                  << " sentences/s" << std::endl;
        timer.start();
      }

      if(batches % 10000 == 0)
        nematus->save("../test/model.marian." + std::to_string(batches) + ".npz");

      batches++;
    }
  }
  std::cout << std::endl;
  std::cout << timer.format(2, "%ws") << std::endl;

  return 0;
}
