#include "hip/hip_runtime.h"

#include "marian.h"
#include "mnist.h"

using namespace marian;
using namespace keywords;

const int input_size = 10;
const int output_size = 15;
const int batch_size = 25;
const int hidden_size = 5;
const int num_inputs = 8;
const int num_outputs = 6;

ExpressionGraph build_graph(int cuda_device) {
  std::cerr << "Building computation graph..." << std::endl;

  ExpressionGraph g(cuda_device);
  std::vector<Expr> X, Y, H, S;

  // For the stop symbol.
  for (int t = 0; t <= num_inputs; ++t) {
    std::stringstream ss;
    ss << "X" << t;
    X.emplace_back(named(g.input(shape={batch_size, input_size}), ss.str()));
  }

  // For the stop symbol.
  for (int t = 0; t <= num_outputs; ++t) {
    std::stringstream ss;
    ss << "Y" << t;
    Y.emplace_back(named(g.input(shape={batch_size, output_size}), ss.str()));
  }

  Expr Wxh = g.param(shape={input_size, hidden_size}, init=uniform(), name="Wxh");
  Expr Whh = g.param(shape={hidden_size, hidden_size}, init=uniform(), name="Whh");
  Expr bh = g.param(shape={1, hidden_size}, init=uniform(), name="bh");
  Expr h0 = g.param(shape={1, hidden_size}, init=uniform(), name="h0");

  std::cerr << "Building encoder RNN..." << std::endl;
  H.emplace_back(tanh(dot(X[0], Wxh) + dot(h0, Whh) + bh));
  for (int t = 1; t <= num_inputs; ++t) {
    H.emplace_back(tanh(dot(X[t], Wxh) + dot(H[t-1], Whh) + bh));
  }

  Expr Wxh_d = g.param(shape={output_size, hidden_size}, init=uniform(), name="Wxh_d");
  Expr Whh_d = g.param(shape={hidden_size, hidden_size}, init=uniform(), name="Whh_d");
  Expr bh_d = g.param(shape={1, hidden_size}, init=uniform(), name="bh_d");

  std::cerr << "Building decoder RNN..." << std::endl;
  auto h0_d = H[num_inputs];
  S.emplace_back(tanh(dot(Y[0], Wxh_d) + dot(h0_d, Whh_d) + bh_d));
  for (int t = 1; t < num_outputs; ++t) {
    S.emplace_back(tanh(dot(Y[t], Wxh_d) + dot(S[t-1], Whh_d) + bh_d));
  }

  Expr Why = g.param(shape={hidden_size, output_size}, init=uniform(), name="Why");
  Expr by = g.param(shape={1, output_size}, init=uniform(), name="by");

  std::cerr << "Building output layer..." << std::endl;
  std::vector<Expr> Yp;

  Yp.emplace_back(named(softmax_fast(dot(h0_d, Why) + by), "pred"));
  Expr cross_entropy = sum(Y[0] * log(Yp[0]), axis=1);
  for (int t = 1; t <= num_outputs; ++t) {
    Yp.emplace_back(named(softmax_fast(dot(S[t-1], Why) + by), "pred"));
    cross_entropy = cross_entropy + sum(Y[t] * log(Yp[t]), axis=1);
  }
  auto graph = -mean(cross_entropy, axis=0, name="cost");

  std::cerr << "Done." << std::endl;

  return g;
}

#if 0
  // read parallel corpus from file
  std::fstream sourceFile("../examples/mt/dev/newstest2013.de");
  std::fstream targetFile("../examples/mt/dev/newstest2013.en");

  std::string sourceLine, targetLine;
  while (getline(sourceFile, sourceLine)) {
    getline(targetFile, targetLine);
    std::vector<size_t> sourceIds = sourceVocab.ProcessSentence(sourceLine);
    std::vector<size_t> targetIds = sourceVocab.ProcessSentence(targetLine);
  }
#endif


int main(int argc, char** argv) {

  ExpressionGraph g = build_graph(0);

  // For the stop symbol.
  for (int t = 0; t <= num_inputs; ++t) {
    Tensor Xt({batch_size, input_size});

    float max = 1.;
    std::vector<float> values(batch_size * input_size);
    std::vector<float> classes(batch_size * output_size, 0.0);
    int k = 0;
    for (int i = 0; i < batch_size; ++i) {
      for (int j = 0; j < input_size; ++j, ++k) {
         values[k] = max * (2.0*static_cast<float>(rand()) / RAND_MAX - 1.0);
      }
    }

    thrust::copy(values.begin(), values.end(), Xt.begin());

    std::stringstream ss;
    ss << "X" << t;
    g[ss.str()] = Xt;

  }

  for (int t = 0; t <= num_outputs; ++t) {
    Tensor Yt({batch_size, output_size});

    std::vector<float> classes(batch_size * output_size, 0.0);
    int l = 0;
    for (int i = 0; i < batch_size; ++i) {
      int gold = output_size * static_cast<float>(rand()) / RAND_MAX;
      classes[l + gold] = 1.0;
      l += output_size;
    }

    thrust::copy(classes.begin(), classes.end(), Yt.begin());

    std::stringstream ss;
    ss << "Y" << t;
    g[ss.str()] = Yt;
  }

  std::cerr << "Graphviz step" << std::endl;
  std::cout << g.graphviz() << std::endl;

  std::cerr << "Forward step" << std::endl;
  g.forward(batch_size);
  std::cerr << "Backward step" << std::endl;
  g.backward();
  std::cerr << "Done" << std::endl;

  std::cerr << g["graph"].val().Debug() << std::endl;

  std::cerr << g["X0"].val().Debug() << std::endl;
  std::cerr << g["Y0"].val().Debug() << std::endl;

#if 0
  std::cerr << Whh.grad().Debug() << std::endl;
  std::cerr << bh.grad().Debug() << std::endl;
  std::cerr << Why.grad().Debug() << std::endl;
  std::cerr << by.grad().Debug() << std::endl;
  std::cerr << Wxh.grad().Debug() << std::endl;
  std::cerr << h0.grad().Debug() << std::endl;
#endif

  return 0;
}
