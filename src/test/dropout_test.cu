#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <vector>
#include <boost/timer/timer.hpp>
#include <boost/chrono.hpp>

#include "training/config.h"
#include "marian.h"
#include "layers/param_initializers.h"

using namespace marian;
using namespace keywords;

int main(int argc, char** argv) {
  auto c = New<Config>(argc, argv);

  auto g = New<ExpressionGraph>();
  g->setDevice(0);
  g->reserveWorkspaceMB(512);

  for(int i = 0; i < 10; ++i) {
    g->clear();
    auto mask = g->dropout(0.2, {10, 3072});
    debug(mask, "mask");
    g->forward();
  }

  return 0;
}
